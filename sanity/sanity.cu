// ====------ sanity.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>


__global__ void test() {
    printf("hello cuda ");
}


int main() {
    test<<<1,1>>>();
    hipblasHandle_t handle;
}