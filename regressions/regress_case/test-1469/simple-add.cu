// ====------ simple-add.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <stdio.h>

const int vector_size = 256;

__global__ void SimpleAddKernel(float *A, int offset) 
{
  A[threadIdx.x] = threadIdx.x + offset;
}

int main() 
{
  float *d_A;
  int offset = 10000;

  hipMalloc( &d_A, vector_size * sizeof( float ) );
  SimpleAddKernel<<<1, vector_size>>>(d_A, offset);

  float result[vector_size] = { };
  hipMemcpy(result, d_A, vector_size*sizeof(float), hipMemcpyDeviceToHost);

  hipFree( d_A );
   
  for (int i = 0; i < vector_size; ++i) {
    if (i % 8 == 0) printf( "\n" );
    printf( "%.1f ", result[i] );
  }

  return 0;
}

