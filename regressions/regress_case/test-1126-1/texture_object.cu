
#include <hip/hip_runtime.h>
// ====------ texture_object.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// Simple transformation kernel
__global__ void transformKernel(float* output, hipTextureObject_t texObj, int width, int height, float theta)
{
// Calculate normalized texture coordinates
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
float u = x / (float)width;
float v = y / (float)height;
// Transform coordinates
u -= 0.5f;
v -= 0.5f;
float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;
// Read from texture and write to global memory
output[y * width + x] = tex2D<float>(texObj, tu, tv);
}

// Host code
int main()
{

float h_data[4][4];
// Allocate CUDA array in device memory
hipChannelFormatDesc channelDesc =
hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
hipArray* cuArray;
int width = 4;
int height = 4;
hipMallocArray(&cuArray, &channelDesc, width, height);
// Copy to device memory some data located at address h_data
// in host memory
int size = width*height*sizeof(float);
hipMemcpyToArray(cuArray, 0, 0, h_data, size,hipMemcpyHostToDevice);
// Specify texture
struct hipResourceDesc resDesc;
memset(&resDesc, 0, sizeof(resDesc));
resDesc.resType = hipResourceTypeArray;
resDesc.res.array.array = cuArray;
// Specify texture object parameters
struct hipTextureDesc texDesc;
memset(&texDesc, 0, sizeof(texDesc));
texDesc.addressMode[0] = hipAddressModeWrap;
texDesc.addressMode[1] = hipAddressModeWrap;
texDesc.filterMode = hipFilterModeLinear;
//texDesc.readMode = cudaReadModeElementType;
texDesc.normalizedCoords = 1;
// Create texture object
hipTextureObject_t texObj = 0;
hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
// Allocate result of transformation in device memory
float* output;
hipMalloc(&output, width * height * sizeof(float));
// Invoke kernel
dim3 dimBlock(height, width);
dim3 dimGrid(1, 1);
transformKernel<<<dimGrid, dimBlock>>>(output, texObj, width, height, 0);
// Destroy texture object
hipDestroyTextureObject(texObj);
// Free device memory
hipFreeArray(cuArray);
hipFree(output);
return 0;
}
