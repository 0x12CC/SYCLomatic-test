// ====------ test.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <assert.h>
 #include <stdlib.h>
 #include <time.h>
#include "hip/hip_runtime.h"
 #include "hiprand/hiprand.h"
 #include "hipblas.h"
 #include <assert.h>
void cuda_random(float *x_gpu, size_t n)
 {
 static hiprandGenerator_t gen[16];
 static int init[16] = {0};
 int i = 0;
 if(!init[i])
{ hiprandCreateGenerator(&gen[i], HIPRAND_RNG_PSEUDO_DEFAULT); hiprandSetPseudoRandomGeneratorSeed(gen[i], time(0)); init[i] = 1; }
hiprandGenerateUniform(gen[i], x_gpu, n);
 }
 
int main() {
return 0;
}
