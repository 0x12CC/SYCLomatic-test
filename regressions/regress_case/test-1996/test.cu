// ====------ test.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>

int main(){
  int val, dev_id;
  
  hipGetDevice(&dev_id);
  hipDeviceAttribute_t attr = hipDeviceAttributeComputeCapabilityMajor;
  hipDeviceGetAttribute(&val, attr, dev_id);

  return 0;

}