// ====------ testThrustTransform.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include<hip/hip_runtime.h>
#include<iostream>



struct my_math
{
  __host__ __device__
  int operator()(const int &r) const{
   return r+1;
  }
};



int main(){


hipStream_t stream;
hipStreamCreate(&stream);

int* host;
hipHostAlloc((void**)&host, 10 * sizeof(int), hipHostMallocDefault);
for(int i=0;i<10;i++)
  host[i]=i;

int *dev_a, *dev_b;
hipMalloc(&dev_a,10*sizeof(int));
hipMalloc(&dev_b,10*sizeof(int));

hipMemcpyAsync(dev_a,host,10*sizeof(int),hipMemcpyHostToDevice,stream);

my_math c;
thrust::transform(thrust::cuda::par.on(stream),dev_a,dev_a + 10,dev_b,c);

hipMemcpyAsync(host,dev_b,10*sizeof(int),hipMemcpyDeviceToHost,stream);

hipStreamSynchronize(stream);
for(int i=0;i<10;i++)
 std::cout<<host[i]<<std::endl;

return 0;


}