// ====------ transfer.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_ERROR_CHECK(x) \
	do { \
		hipError_t last_err = (x); \
		if (last_err != hipSuccess) { \
			fprintf(stderr, "%s:%u: CUDA error: %s\n", __FILE__, __LINE__, \
					hipGetErrorString(last_err)); \
			exit(1); \
		} \
	} while (false)

#define CUDA_CALL(x) CUDA_ERROR_CHECK(x)

int main(void)
{  
	const size_t buf_count = 1024;
	const size_t buf_size = buf_count * sizeof(double);

	double *buf = (double*) calloc(buf_size, buf_size);
	double *buf_dev;

	CUDA_CALL(hipMalloc((void**)&buf_dev, buf_size)); 
	CUDA_CALL(hipMemcpy(buf_dev, buf, buf_size, hipMemcpyHostToDevice));
	
	CUDA_CALL(hipDeviceSynchronize());
	
	CUDA_CALL(hipMemcpy(buf, buf_dev, buf_size, hipMemcpyDeviceToHost));
	
	CUDA_CALL(hipFree(buf_dev));
	free(buf);
	
	return EXIT_SUCCESS;
}
