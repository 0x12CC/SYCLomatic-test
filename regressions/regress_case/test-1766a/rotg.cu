// ====------ rotg.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

bool foo1(){
  float a = 1.0f;
  float b = 1.0f;
  float c;
  float s;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  hipblasSrotg(handle, &a, &b, &c, &s);
  hipblasDestroy(handle);
  if ((std::abs(a - 1.41421) < 0.01) && (std::abs(b - 1.41421) < 0.01) &&
      (std::abs(c - 0.707107) < 0.01) && (std::abs(s - 0.707107) < 0.01)) {
    return true;
  } else {
    std::cout << "foo1() failed" << std::endl;
    return false;
  }
}

bool foo2(){
  float a = 1.0f;
  float b = 1.0f;
  float c;
  float s;

  float *d_a, *d_b, *d_c, *d_s;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipMalloc(&d_a, sizeof(float));
  hipMalloc(&d_b, sizeof(float));
  hipMalloc(&d_c, sizeof(float));
  hipMalloc(&d_s, sizeof(float));

  hipMemcpy(d_a, &a, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, sizeof(float), hipMemcpyHostToDevice);
  hipblasSrotg(handle, d_a, d_b, d_c, d_s);
  hipDeviceSynchronize();

  hipMemcpy(&a, d_a, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&b, d_b, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&c, d_c, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&s, d_s, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_s);
  hipblasDestroy(handle);

  if ((std::abs(a - 1.41421) < 0.01) && (std::abs(b - 1.41421) < 0.01) &&
      (std::abs(c - 0.707107) < 0.01) && (std::abs(s - 0.707107) < 0.01)) {
    return true;
  } else {
    std::cout << "foo2() failed" << std::endl;
    return false;
  }
}

bool foo3(){
  double a = 1.0;
  double b = 1.0;
  double c;
  double s;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  hipblasDrotg(handle, &a, &b, &c, &s);
  hipblasDestroy(handle);
  if ((std::abs(a - 1.41421) < 0.01) && (std::abs(b - 1.41421) < 0.01) &&
      (std::abs(c - 0.707107) < 0.01) && (std::abs(s - 0.707107) < 0.01)) {
    return true;
  } else {
    std::cout << "foo3() failed" << std::endl;
    return false;
  }
}

bool foo4(){
  double a = 1.0;
  double b = 1.0;
  double c;
  double s;

  double *d_a, *d_b, *d_c, *d_s;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipMalloc(&d_a, sizeof(double));
  hipMalloc(&d_b, sizeof(double));
  hipMalloc(&d_c, sizeof(double));
  hipMalloc(&d_s, sizeof(double));

  hipMemcpy(d_a, &a, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, sizeof(double), hipMemcpyHostToDevice);
  hipblasDrotg(handle, d_a, d_b, d_c, d_s);
  hipDeviceSynchronize();

  hipMemcpy(&a, d_a, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&b, d_b, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&c, d_c, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&s, d_s, sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_s);
  hipblasDestroy(handle);

  if ((std::abs(a - 1.41421) < 0.01) && (std::abs(b - 1.41421) < 0.01) &&
      (std::abs(c - 0.707107) < 0.01) && (std::abs(s - 0.707107) < 0.01)) {
    return true;
  } else {
    std::cout << "foo4() failed" << std::endl;
    return false;
  }
}

bool foo5(){
  float2 a;
  a.x = 1.0f;
  a.y = 1.0f;
  float2 b;
  b.x = 1.0f;
  b.y = 1.0f;
  float c;
  float2 s;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  hipblasCrotg(handle, &a, &b, &c, &s);
  hipblasDestroy(handle);

  if ((std::abs(a.x - 1.41421) < 0.01) && (std::abs(a.y - 1.41421) < 0.01) &&
      (std::abs(b.x - 1) < 0.01) && (std::abs(b.y - 1) < 0.01)  &&
      (std::abs(c - 0.707107) < 0.01) && (std::abs(s.x - 0.707107) < 0.01) &&
      (std::abs(s.y) < 0.01)) {
    return true;
  } else {
    std::cout << "foo5() failed" << std::endl;
    return false;
  }
}

bool foo6(){
  float2 a;
  a.x = 1.0;
  a.y = 1.0;
  float2 b;
  b.x = 1.0;
  b.y = 1.0;
  float c;
  float2 s;

  float2 *d_a, *d_b, *d_s;
  float *d_c;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipMalloc(&d_a, sizeof(float2));
  hipMalloc(&d_b, sizeof(float2));
  hipMalloc(&d_c, sizeof(float));
  hipMalloc(&d_s, sizeof(float2));

  hipMemcpy(d_a, &a, sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, sizeof(float2), hipMemcpyHostToDevice);
  hipblasCrotg(handle, d_a, d_b, d_c, d_s);
  hipDeviceSynchronize();


  hipMemcpy(&a, d_a, sizeof(float2), hipMemcpyDeviceToHost);
  hipMemcpy(&b, d_b, sizeof(float2), hipMemcpyDeviceToHost);
  hipMemcpy(&c, d_c, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&s, d_s, sizeof(float2), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_s);

  hipblasDestroy(handle);

  if ((std::abs(a.x - 1.41421) < 0.01) && (std::abs(a.y - 1.41421) < 0.01) &&
      (std::abs(b.x - 1) < 0.01) && (std::abs(b.y - 1) < 0.01)  &&
      (std::abs(c - 0.707107) < 0.01) && (std::abs(s.x - 0.707107) < 0.01) &&
      (std::abs(s.y) < 0.01)) {
    return true;
  } else {
    std::cout << "foo6() failed" << std::endl;
    return false;
  }
}

bool foo7(){
  double2 a;
  a.x = 1.0;
  a.y = 1.0;
  double2 b;
  b.x = 1.0;
  b.y = 1.0;
  double c;
  double2 s;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  hipblasZrotg(handle, &a, &b, &c, &s);
  hipblasDestroy(handle);

  if ((std::abs(a.x - 1.41421) < 0.01) && (std::abs(a.y - 1.41421) < 0.01) &&
      (std::abs(b.x - 1) < 0.01) && (std::abs(b.y - 1) < 0.01)  &&
      (std::abs(c - 0.707107) < 0.01) && (std::abs(s.x - 0.707107) < 0.01) &&
      (std::abs(s.y) < 0.01)) {
    return true;
  } else {
    std::cout << "foo7() failed" << std::endl;
    return false;
  }
}

bool foo8(){
  double2 a;
  a.x = 1.0;
  a.y = 1.0;
  double2 b;
  b.x = 1.0;
  b.y = 1.0;
  double c;
  double2 s;

  double2 *d_a, *d_b, *d_s;
  double *d_c;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipMalloc(&d_a, sizeof(double2));
  hipMalloc(&d_b, sizeof(double2));
  hipMalloc(&d_c, sizeof(double));
  hipMalloc(&d_s, sizeof(double2));

  hipMemcpy(d_a, &a, sizeof(double2), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, sizeof(double2), hipMemcpyHostToDevice);
  hipblasZrotg(handle, d_a, d_b, d_c, d_s);
  hipDeviceSynchronize();


  hipMemcpy(&a, d_a, sizeof(double2), hipMemcpyDeviceToHost);
  hipMemcpy(&b, d_b, sizeof(double2), hipMemcpyDeviceToHost);
  hipMemcpy(&c, d_c, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&s, d_s, sizeof(double2), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_s);

  hipblasDestroy(handle);

  if ((std::abs(a.x - 1.41421) < 0.01) && (std::abs(a.y - 1.41421) < 0.01) &&
      (std::abs(b.x - 1) < 0.01) && (std::abs(b.y - 1) < 0.01)  &&
      (std::abs(c - 0.707107) < 0.01) && (std::abs(s.x - 0.707107) < 0.01) &&
      (std::abs(s.y) < 0.01)) {
    return true;
  } else {
    std::cout << "foo8() failed" << std::endl;
    return false;
  }
}

bool foo9(){
  float d1 = 1.0f;
  float d2 = 4.0f;
  float x1 = 1.0f;
  float y1 = 1.0f;
  float *param = (float *)malloc(5 * sizeof(float));

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  hipblasSrotmg(handle, &d1, &d2, &x1, &y1, param);
  hipblasDestroy(handle);

  if ((std::abs(d1 - 3.2) < 0.01) && (std::abs(d2 - 0.8) < 0.01) &&
      (std::abs(x1 - 1.25) < 0.01) && (std::abs(param[0] - 1) < 0.01)  &&
      (std::abs(param[1] - 0.25) < 0.01) && (std::abs(param[4] - 1) < 0.01)) {
    free(param);
    return true;
  } else {
    free(param);
    std::cout << "foo9() failed" << std::endl;
    return false;
  }
}

bool foo10(){
  float d1 = 1.0f;
  float d2 = 4.0f;
  float x1 = 1.0f;
  float y1 = 1.0f;
  float *param = (float *)malloc(5 * sizeof(float));

  float *d_d1, *d_d2, *d_x1, *d_y1, *d_param;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipMalloc(&d_d1, sizeof(float));
  hipMalloc(&d_d2, sizeof(float));
  hipMalloc(&d_x1, sizeof(float));
  hipMalloc(&d_y1, sizeof(float));
  hipMalloc(&d_param, 5 * sizeof(float));

  hipMemcpy(d_d1, &d1, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_d2, &d2, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_x1, &x1, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y1, &y1, sizeof(float), hipMemcpyHostToDevice);
  hipblasSrotmg(handle, d_d1, d_d2, d_x1, d_y1, d_param);
  hipDeviceSynchronize();

  hipMemcpy(&d1, d_d1, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&d2, d_d2, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&x1, d_x1, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(param, d_param, 5 * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_d1);
  hipFree(d_d2);
  hipFree(d_x1);
  hipFree(d_y1);
  hipFree(d_param);
  hipblasDestroy(handle);

  if ((std::abs(d1 - 3.2) < 0.01) && (std::abs(d2 - 0.8) < 0.01) &&
      (std::abs(x1 - 1.25) < 0.01) && (std::abs(param[0] - 1) < 0.01)  &&
      (std::abs(param[1] - 0.25) < 0.01) && (std::abs(param[4] - 1) < 0.01)) {
    free(param);
    return true;
  } else {
    free(param);
    std::cout << "foo10() failed" << std::endl;
    return false;
  }
}

bool foo11(){
  double d1 = 1.0;
  double d2 = 4.0;
  double x1 = 1.0;
  double y1 = 1.0;
  double *param = (double *)malloc(5 * sizeof(double));

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  hipblasDrotmg(handle, &d1, &d2, &x1, &y1, param);
  hipblasDestroy(handle);

  if ((std::abs(d1 - 3.2) < 0.01) && (std::abs(d2 - 0.8) < 0.01) &&
      (std::abs(x1 - 1.25) < 0.01) && (std::abs(param[0] - 1) < 0.01)  &&
      (std::abs(param[1] - 0.25) < 0.01) && (std::abs(param[4] - 1) < 0.01)) {
    free(param);
    return true;
  } else {
    free(param);
    std::cout << "foo11() failed" << std::endl;
    return false;
  }
}

bool foo12(){
  double d1 = 1.0;
  double d2 = 4.0;
  double x1 = 1.0;
  double y1 = 1.0;
  double *param = (double *)malloc(5 * sizeof(double));

  double *d_d1, *d_d2, *d_x1, *d_y1, *d_param;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipMalloc(&d_d1, sizeof(double));
  hipMalloc(&d_d2, sizeof(double));
  hipMalloc(&d_x1, sizeof(double));
  hipMalloc(&d_y1, sizeof(double));
  hipMalloc(&d_param, 5 * sizeof(double));

  hipMemcpy(d_d1, &d1, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_d2, &d2, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_x1, &x1, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_y1, &y1, sizeof(double), hipMemcpyHostToDevice);
  hipblasDrotmg(handle, d_d1, d_d2, d_x1, d_y1, d_param);
  hipDeviceSynchronize();

  hipMemcpy(&d1, d_d1, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&d2, d_d2, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&x1, d_x1, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(param, d_param, 5 * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_d1);
  hipFree(d_d2);
  hipFree(d_x1);
  hipFree(d_y1);
  hipFree(d_param);
  hipblasDestroy(handle);

  if ((std::abs(d1 - 3.2) < 0.01) && (std::abs(d2 - 0.8) < 0.01) &&
      (std::abs(x1 - 1.25) < 0.01) && (std::abs(param[0] - 1) < 0.01)  &&
      (std::abs(param[1] - 0.25) < 0.01) && (std::abs(param[4] - 1) < 0.01)) {
    free(param);
    return true;
  } else {
    free(param);
    std::cout << "foo12() failed" << std::endl;
    return false;
  }
}

int main(){
  if (foo1() && foo2() && foo3() && foo4() &&
      foo5() && foo6() && foo7() && foo8() &&
      foo9() && foo10() && foo11() && foo12()) {
    std::cout << "pass" << std::endl;
    return 0;
  } else {
    std::cout << "fail" << std::endl;
    return 1;
  }
}

