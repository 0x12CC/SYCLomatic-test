// ====------ cublas-create-Sgemm-destroy.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

extern hipblasHandle_t handle2;

int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }
  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);



  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}
