// ====------ cusolverDnLn.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    status = hipsolverDnSpotrf_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotrf_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotrf_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotrf_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    status = hipsolverDnSgetrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnDgetrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnCgetrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnZgetrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);

    status = hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);


    status = hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);
    hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);

    status = hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);
    hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);

    status = hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);
    hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);


    status = hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);
    hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);

    status = hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);
    hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);

    status = hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);
    hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);

    status = hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);
    hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);

    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    status = hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);
    hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);

    status = hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);
    hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);

    status = hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);
    hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);

    status = hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);
    hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);

    status = hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    status = hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    status = hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    status = hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    status = hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);


    status = hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    status = hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    status = hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    status = hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    status = hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    status = hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    status = hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    status = hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);
    hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    status = hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
    hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
}
