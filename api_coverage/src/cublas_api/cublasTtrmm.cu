// ====------ cublasTtrmm.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;
  const float *A_S = 0;
  const float *B_S = 0;
  float *C_S = 0;
  float alpha_S = 1.0f;
  const double *A_D = 0;
  const double *B_D = 0;
  double *C_D = 0;
  double alpha_D = 1.0;

  int side0 = 0; int side1 = 1; int fill0 = 0; int fill1 = 1;
  int trans0 = 0; int trans1 = 1; int trans2 = 2; int diag0 = 0; int diag1 = 1;
  status = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_S, A_S, lda, B_S, ldb, C_S, ldc);
  hipblasStrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_S, A_S, lda, B_S, ldb, C_S, ldc);


  status = hipblasDtrmm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill1, (hipblasOperation_t)trans1, (hipblasDiagType_t)diag1, m, n, &alpha_D, A_D, lda, B_D, ldb, C_D, ldc);
  hipblasDtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_D, A_D, lda, B_D, ldb, C_D, ldc);


  const hipComplex *A_C = 0;
  const hipComplex *B_C = 0;
  hipComplex *C_C = 0;
  hipComplex alpha_C = make_hipComplex(1.0f,0.0f);
  const hipDoubleComplex *A_Z = 0;
  const hipDoubleComplex *B_Z = 0;
  hipDoubleComplex *C_Z = 0;
  hipDoubleComplex alpha_Z = make_hipDoubleComplex(1.0,0.0);


  status = hipblasCtrmm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)0, (hipblasOperation_t)trans2, (hipblasDiagType_t)0, m, n, &alpha_C, A_C, lda, B_C, ldb, C_C, ldc);
  hipblasCtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_C, A_C, lda, B_C, ldb, C_C, ldc);


  status = hipblasZtrmm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)1, (hipblasOperation_t)2, (hipblasDiagType_t)1, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, C_Z, ldc);
  hipblasZtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, C_Z, ldc);

}
