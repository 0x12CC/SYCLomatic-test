// ====------ cublasTsyrkx.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define macro_a (hipblasOperation_t)1

#define macro_b (hipblasFillMode_t)1

hipblasFillMode_t foo(){
  return HIPBLAS_FILL_MODE_LOWER;
}

hipblasOperation_t bar(){
  return HIPBLAS_OP_T;
}

int main() {
  int n = 275;
  int k = 275;
  int lda = 1;
  int ldb = 1;
  int ldc = 1;

  float alpha_s = 1;
  float beta_s = 1;

  double alpha_d = 1;
  double beta_d = 1;

  hipblasHandle_t handle;
  hipblasStatus_t status;

  float* A_s=0;
  float* B_s=0;
  float* C_s=0;

  double* A_d=0;
  double* B_d=0;
  double* C_d=0;

  int trans0 = 0;
  int trans1 = 1;
  int fill0 = 0;
  int fill1 = 1;

  status = hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);
  hipblasSsyrkx(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)trans1, n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);

  status = hipblasDsyrkx(handle, (hipblasFillMode_t)0, (hipblasOperation_t)0, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);
  hipblasDsyrkx(handle, (hipblasFillMode_t)1, (hipblasOperation_t)1, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);




  hipblasDsyrkx(handle, foo(), macro_a, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);


  hipblasDsyrkx(handle, macro_b, bar(), n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);


  hipblasDsyrkx(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);

  return 0;
}
