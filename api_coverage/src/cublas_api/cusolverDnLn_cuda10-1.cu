// ====------ cusolverDnLn_cuda10-1.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.2, v10.0
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    int m = 0;
    int n = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);
    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;

    status = hipsolverDnSpotri_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotri_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotri_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotri_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);


    status = hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
}
