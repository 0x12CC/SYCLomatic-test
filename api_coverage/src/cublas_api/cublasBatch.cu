// ====------ cublasBatch.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int nrhs = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;

  float **Aarray_S = 0;
  double **Aarray_D = 0;
  hipComplex **Aarray_C = 0;
  hipDoubleComplex **Aarray_Z = 0;

  float **Barray_S = 0;
  double **Barray_D = 0;
  hipComplex **Barray_C = 0;
  hipDoubleComplex **Barray_Z = 0;

  float **Carray_S = 0;
  double **Carray_D = 0;
  hipComplex **Carray_C = 0;
  hipDoubleComplex **Carray_Z = 0;

  float **TauArray_S = 0;
  double **TauArray_D = 0;
  hipComplex **TauArray_C = 0;
  hipDoubleComplex **TauArray_Z = 0;

  int *PivotArray = 0;
  int *infoArray = 0;
  int batchSize = 10;

  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;

  const float **Aarray_Sc = 0;
  const double **Aarray_Dc = 0;
  const hipComplex **Aarray_Cc = 0;
  const hipDoubleComplex **Aarray_Zc = 0;

  // getrf_batch
  status = hipblasSgetrfBatched(handle, n, Aarray_S, lda, PivotArray, infoArray, batchSize);
  hipblasSgetrfBatched(handle, n, Aarray_S, lda, PivotArray, infoArray, batchSize);

  status = hipblasDgetrfBatched(handle, n, Aarray_D, lda, PivotArray, infoArray, batchSize);
  hipblasDgetrfBatched(handle, n, Aarray_D, lda, PivotArray, infoArray, batchSize);

  status = hipblasCgetrfBatched(handle, n, Aarray_C, lda, PivotArray, infoArray, batchSize);
  hipblasCgetrfBatched(handle, n, Aarray_C, lda, PivotArray, infoArray, batchSize);

  status = hipblasZgetrfBatched(handle, n, Aarray_Z, lda, PivotArray, infoArray, batchSize);
  hipblasZgetrfBatched(handle, n, Aarray_Z, lda, PivotArray, infoArray, batchSize);

  // getrs_batch
  status = hipblasSgetrsBatched(handle, (hipblasOperation_t)trans0, n, nrhs, Aarray_Sc, lda, PivotArray, Barray_S, ldb, infoArray, batchSize);
  hipblasSgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Sc, lda, PivotArray, Barray_S, ldb, infoArray, batchSize);

  status = hipblasDgetrsBatched(handle, (hipblasOperation_t)trans1, n, nrhs, Aarray_Dc, lda, PivotArray, Barray_D, ldb, infoArray, batchSize);
  hipblasDgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Dc, lda, PivotArray, Barray_D, ldb, infoArray, batchSize);

  status = hipblasCgetrsBatched(handle, (hipblasOperation_t)trans2, n, nrhs, Aarray_Cc, lda, PivotArray, Barray_C, ldb, infoArray, batchSize);
  hipblasCgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Cc, lda, PivotArray, Barray_C, ldb, infoArray, batchSize);

  status = hipblasZgetrsBatched(handle, (hipblasOperation_t)2, n, nrhs, Aarray_Zc, lda, PivotArray, Barray_Z, ldb, infoArray, batchSize);
  hipblasZgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Zc, lda, PivotArray, Barray_Z, ldb, infoArray, batchSize);

  // getri_batch
  status = hipblasSgetriBatched(handle, n, Aarray_Sc, lda, PivotArray, Carray_S, ldc, infoArray, batchSize);
  hipblasSgetriBatched(handle, n, Aarray_Sc, lda, PivotArray, Carray_S, ldc, infoArray, batchSize);

  status = hipblasDgetriBatched(handle, n, Aarray_Dc, lda, PivotArray, Carray_D, ldc, infoArray, batchSize);
  hipblasDgetriBatched(handle, n, Aarray_Dc, lda, PivotArray, Carray_D, ldc, infoArray, batchSize);

  status = hipblasCgetriBatched(handle, n, Aarray_Cc, lda, PivotArray, Carray_C, ldc, infoArray, batchSize);
  hipblasCgetriBatched(handle, n, Aarray_Cc, lda, PivotArray, Carray_C, ldc, infoArray, batchSize);

  status = hipblasZgetriBatched(handle, n, Aarray_Zc, lda, PivotArray, Carray_Z, ldc, infoArray, batchSize);
  hipblasZgetriBatched(handle, n, Aarray_Zc, lda, PivotArray, Carray_Z, ldc, infoArray, batchSize);

  // geqrf_batch
  status = hipblasSgeqrfBatched(handle, m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize);
  hipblasSgeqrfBatched(handle, m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize);

  status = hipblasDgeqrfBatched(handle, m, n, Aarray_D, lda, TauArray_D, infoArray, batchSize);
  hipblasDgeqrfBatched(handle, m, n, Aarray_D, lda, TauArray_D, infoArray, batchSize);

  status = hipblasCgeqrfBatched(handle, m, n, Aarray_C, lda, TauArray_C, infoArray, batchSize);
  hipblasCgeqrfBatched(handle, m, n, Aarray_C, lda, TauArray_C, infoArray, batchSize);

  status = hipblasZgeqrfBatched(handle, m, n, Aarray_Z, lda, TauArray_Z, infoArray, batchSize);
  hipblasZgeqrfBatched(handle, m, n, Aarray_Z, lda, TauArray_Z, infoArray, batchSize);

  return 0;
}
