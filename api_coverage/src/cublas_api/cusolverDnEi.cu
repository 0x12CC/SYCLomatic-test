// ====------ cusolverDnEi.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    hipsolverEigMode_t jobz;

    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    signed char jobu;
    signed char jobvt;

    float S_f = 0;
    double S_d = 0.0;
    hipComplex S_c = make_hipComplex(1,0);
    hipDoubleComplex S_z = make_hipDoubleComplex(1,0);

    float U_f = 0;
    double U_d = 0.0;
    hipComplex U_c = make_hipComplex(1,0);
    hipDoubleComplex U_z = make_hipDoubleComplex(1,0);
    int ldu;

    float VT_f = 0;
    double VT_d = 0.0;
    hipComplex VT_c = make_hipComplex(1,0);
    hipDoubleComplex VT_z = make_hipDoubleComplex(1,0);
    int ldvt;

    float Rwork_f = 0;
    double Rwork_d = 0.0;
    hipComplex Rwork_c = make_hipComplex(1,0);
    hipDoubleComplex Rwork_z = make_hipDoubleComplex(1,0);

    float W_f = 0;
    double W_d = 0.0;
    hipComplex W_c = make_hipComplex(1,0);
    hipDoubleComplex W_z = make_hipDoubleComplex(1,0);

    status = hipsolverDnSgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnSgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnDgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnCgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnZgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);


    status = hipsolverDnSorgbr_bufferSize(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgbr_bufferSize(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);


    status = hipsolverDnDorgbr_bufferSize(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgbr_bufferSize(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCungbr_bufferSize(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungbr_bufferSize(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZungbr_bufferSize(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungbr_bufferSize(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);


    status = hipsolverDnSsytrd_bufferSize(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &Lwork);
    hipsolverDnSsytrd_bufferSize(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &Lwork);
    status = hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDsytrd_bufferSize(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &Lwork);
    hipsolverDnDsytrd_bufferSize(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &Lwork);
    status = hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnChetrd_bufferSize(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &Lwork);
    hipsolverDnChetrd_bufferSize(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &Lwork);
    status = hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZhetrd_bufferSize(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &Lwork);
    hipsolverDnZhetrd_bufferSize(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &Lwork);
    status = hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);

    status = hipsolverDnSormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &Lwork);
    hipsolverDnSormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &Lwork);
    status = hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &Lwork);
    hipsolverDnDormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &Lwork);
    status = hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &Lwork);
    hipsolverDnCunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &Lwork);
    status = hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &Lwork);
    hipsolverDnZunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &Lwork);
    status = hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    status = hipsolverDnSorgtr_bufferSize(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgtr_bufferSize(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    status = hipsolverDnDorgtr_bufferSize(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgtr_bufferSize(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    status = hipsolverDnCungtr_bufferSize(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungtr_bufferSize(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    status = hipsolverDnZungtr_bufferSize(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungtr_bufferSize(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);


    status = hipsolverDnSgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnSgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);
    hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);

    status = hipsolverDnDgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnDgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);
    hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);

    status = hipsolverDnCgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnCgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);
    hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);

    status = hipsolverDnZgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnZgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);
    hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);

}
