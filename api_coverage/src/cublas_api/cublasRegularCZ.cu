// ====------ cublasRegularCZ.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;

  int* result = 0;
  float* result_f = 0;
  double* result_d = 0;
  hipComplex* x_c = 0;
  hipDoubleComplex* x_z = 0;

  int incx = 1;
  int incy = 1;
  int n = 10;

  //level 1
  status = hipblasIcamax(handle, n, x_c, incx, result);
  hipblasIcamax(handle, n, x_c, incx, result);

  status = hipblasIzamax(handle, n, x_z, incx, result);
  hipblasIzamax(handle, n, x_z, incx, result);

  status = hipblasIcamin(handle, n, x_c, incx, result);
  hipblasIcamin(handle, n, x_c, incx, result);

  status = hipblasIzamin(handle, n, x_z, incx, result);
  hipblasIzamin(handle, n, x_z, incx, result);

  status = hipblasScasum(handle, n, x_c, incx, result_f);
  hipblasScasum(handle, n, x_c, incx, result_f);

  status = hipblasDzasum(handle, n, x_z, incx, result_d);
  hipblasDzasum(handle, n, x_z, incx, result_d);

  hipComplex* alpha_c = 0;
  hipComplex* beta_c = 0;
  hipDoubleComplex* alpha_z = 0;
  hipDoubleComplex* beta_z = 0;
  float* alpha_f = 0;
  double* alpha_d = 0;
  hipComplex* y_c = 0;
  hipDoubleComplex* y_z = 0;

  status = hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);
  hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);

  status = hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);
  hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);

  status = hipblasCcopy(handle, n, x_c, incx, y_c, incy);
  hipblasCcopy(handle, n, x_c, incx, y_c, incy);

  status = hipblasZcopy(handle, n, x_z, incx, y_z, incy);
  hipblasZcopy(handle, n, x_z, incx, y_z, incy);

  hipComplex* result_c = 0;
  hipDoubleComplex* result_z = 0;

  status = hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);

  status = hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);

  status = hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);

  status = hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);

  status = hipblasScnrm2(handle, n, x_c, incx, result_f);
  hipblasScnrm2(handle, n, x_c, incx, result_f);

  status = hipblasDznrm2(handle, n, x_z, incx, result_d);
  hipblasDznrm2(handle, n, x_z, incx, result_d);

  float* c_f = 0;
  float* s_f = 0;
  double* c_d = 0;
  double* s_d = 0;
  hipComplex* c_c = 0;
  hipComplex* s_c = 0;
  hipDoubleComplex* c_z = 0;
  hipDoubleComplex* s_z = 0;

  status = hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);
  hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);

  status = hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);
  hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);

  status = hipblasCrotg(handle, x_c, y_c, c_f, s_c);
  hipblasCrotg(handle, x_c, y_c, c_f, s_c);

  status = hipblasZrotg(handle, x_z, y_z, c_d, s_z);
  hipblasZrotg(handle, x_z, y_z, c_d, s_z);

  status = hipblasCscal(handle, n, alpha_c, x_c, incx);
  hipblasCscal(handle, n, alpha_c, x_c, incx);

  status = hipblasZscal(handle, n, alpha_z, x_z, incx);
  hipblasZscal(handle, n, alpha_z, x_z, incx);

  status = hipblasCsscal(handle, n, alpha_f, x_c, incx);
  hipblasCsscal(handle, n, alpha_f, x_c, incx);

  status = hipblasZdscal(handle, n, alpha_d, x_z, incx);
  hipblasZdscal(handle, n, alpha_d, x_z, incx);

  status = hipblasCswap(handle, n, x_c, incx, y_c, incy);
  hipblasCswap(handle, n, x_c, incx, y_c, incy);

  status = hipblasZswap(handle, n, x_z, incx, y_z, incy);
  hipblasZswap(handle, n, x_z, incx, y_z, incy);

  //level 2
  int m=0;
  int kl=0;
  int ku=0;
  int lda = 10;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  status = hipblasCgbmv(handle, (hipblasOperation_t)trans0, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  status = hipblasZgbmv(handle, (hipblasOperation_t)trans1, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  status = hipblasCgemv(handle, (hipblasOperation_t)trans2, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgemv(handle, HIPBLAS_OP_N, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  status = hipblasZgemv(handle, (hipblasOperation_t)0, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  status = hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  status = hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  status = hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  status = hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  int k = 1;
  int fill0 = 0;
  int fill1 = 1;
  int diag0 = 0;
  int diag1 = 1;
  status = hipblasCtbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)1, (hipblasDiagType_t)diag0, n, k, x_c, lda, result_c, incx);
  hipblasCtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_c, lda, result_c, incx);

  status = hipblasZtbmv(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)2, (hipblasDiagType_t)diag1, n, k, x_z, lda, result_z, incx);
  hipblasZtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_z, lda, result_z, incx);

  status = hipblasCtbsv(handle, (hipblasFillMode_t)0, (hipblasOperation_t)trans0, (hipblasDiagType_t)0,  n, k, x_c, lda, result_c, incx);
  hipblasCtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_c, lda, result_c, incx);

  status = hipblasZtbsv(handle, (hipblasFillMode_t)1, (hipblasOperation_t)trans0, (hipblasDiagType_t)1,  n, k, x_z, lda, result_z, incx);
  hipblasZtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_z, lda, result_z, incx);

  status = hipblasCtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  status = hipblasZtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  status = hipblasCtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  status = hipblasZtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  status = hipblasCtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  status = hipblasZtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  status = hipblasCtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  status = hipblasZtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  status = hipblasChemv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  status = hipblasZhemv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  status = hipblasChbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  status = hipblasZhbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  status = hipblasChpmv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);
  hipblasChpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);

  status = hipblasZhpmv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);
  hipblasZhpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);

  status = hipblasCher(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c, lda);
  hipblasCher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c, lda);

  status = hipblasZher(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z, lda);
  hipblasZher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z, lda);

  status = hipblasCher2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  status = hipblasZher2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  status = hipblasChpr(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c);
  hipblasChpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c);

  status = hipblasZhpr(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z);
  hipblasZhpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z);

  status = hipblasChpr2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c);
  hipblasChpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c);

  status = hipblasZhpr2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z);
  hipblasZhpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z);

  int N = 100;
  status = hipblasCgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);

  status = hipblasZgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);

  hipComplex* A_c = 0;
  hipDoubleComplex* A_z = 0;
  hipComplex* B_c = 0;
  hipDoubleComplex* B_z = 0;
  hipComplex* C_c = 0;
  hipDoubleComplex* C_z = 0;


  int ldb = 10;
  int ldc = 10;


  const float alpha_s = 1;
  const float beta_s = 1;
  const double beta_d = 0;



  int side0 = 0;
  int side1 = 1;
  status = hipblasCsymm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  status = hipblasZsymm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasCsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  hipblasCsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);

  status = hipblasZsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);
  hipblasZsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  status = hipblasCsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  status = hipblasZsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasCtrsm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_c, A_c, lda, B_c, ldb);
  hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_c, A_c, lda, B_c, ldb);

  status = hipblasZtrsm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_z, A_z, lda, B_z, ldb);
  hipblasZtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_z, A_z, lda, B_z, ldb);

  status = hipblasChemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasChemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  status = hipblasZhemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZhemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasCherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);
  hipblasCherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);

  status = hipblasZherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);
  hipblasZherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);

  status = hipblasCher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);
  hipblasCher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);

  status = hipblasZher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);
  hipblasZher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);
}
