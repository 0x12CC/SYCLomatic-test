// ====------ test.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void hello_world(void)
{
  printf("GPU: Hello world!\n");
}
int main(int argc,char **argv)
{
  printf("CPU: Hello world!\n");
  hello_world<<<1,10>>>();
  hipDeviceReset();//if no this line ,it can not output hello world from gpu
  return 0;
}


