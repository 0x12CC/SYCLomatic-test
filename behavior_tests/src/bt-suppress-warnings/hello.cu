// ====------ hello.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <time.h>

void test() {
  hipError_t err;
  int i = 0;
  if (err != hipSuccess) {
    ++i;
  }

  if (err == hipErrorAssert) {
    printf("efef");
    malloc(0x100);
  }
}

int main() {
  float *a;
  int r = hipMalloc((void**)&a, sizeof(float));

  clock_t timer = clock();
}
