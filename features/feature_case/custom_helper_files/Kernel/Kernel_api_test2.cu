
#include <hip/hip_runtime.h>
// ====------ Kernel_api_test2.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Kernel/api_test2_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Kernel/api_test2_out/MainSourceFiles.yaml | wc -l > %T/Kernel/api_test2_out/count.txt
// RUN: FileCheck --input-file %T/Kernel/api_test2_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Kernel/api_test2_out

// CHECK: 16
// TEST_FEATURE: Kernel_get_kernel_function_info

__global__ void foo() {}

int main() {
  hipFuncAttributes attrs;
  hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(foo));
  return 0;
}
