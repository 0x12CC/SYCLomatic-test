
#include <hip/hip_runtime.h>
// ====------ Image_api_test18.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test18_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test18_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test18_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test18_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test18_out

// CHECK: 30
// TEST_FEATURE: Image_image_wrapper_base_set_channel_type
// TEST_FEATURE: Image_image_wrapper_base_set_channel_num

int main() {
  hipTexRef tex;
  hipArray_Format format;
  hipTexRefSetFormat(tex, format, 4);
  return 0;
}
