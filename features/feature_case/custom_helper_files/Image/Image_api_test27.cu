
#include <hip/hip_runtime.h>
// ====------ Image_api_test27.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test27_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test27_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test27_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test27_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test27_out

// CHECK: 21
// TEST_FEATURE: Image_image_data_set_data_type

int main() {
  hipResourceDesc res21;
  res21.resType = hipResourceTypeLinear;
  return 0;
}
