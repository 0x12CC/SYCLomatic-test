
#include <hip/hip_runtime.h>
// ====------ Image_api_test20.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test20_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test20_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test20_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test20_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test20_out

// CHECK: 19
// TEST_FEATURE: Image_image_matrix_get_range
// TEST_FEATURE: Image_image_matrix_get_range_T

int main() {
  hipChannelFormatDesc desc;
  hipExtent extent = make_hipExtent(1, 1, 1);
  unsigned int flags;
  hipArray_t array;

  hipArrayGetInfo(&desc, &extent, &flags, array);
  return 0;
}
