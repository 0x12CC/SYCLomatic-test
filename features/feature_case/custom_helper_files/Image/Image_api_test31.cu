
#include <hip/hip_runtime.h>
// ====------ Image_api_test31.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test31_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test31_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test31_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test31_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test31_out

// CHECK: 30
// TEST_FEATURE: Image_image_channel_set_channel_data_type
// TEST_FEATURE: Image_image_channel_set_channel_size

int main() {
  hipChannelFormatDesc Ch;
  Ch.w = 123;
  hipChannelFormatKind type;
  Ch.f = type;
  return 0;
}
