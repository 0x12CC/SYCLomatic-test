
#include <hip/hip_runtime.h>
// ====------ Image_api_test28.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test28_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test28_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test28_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test28_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test28_out

// CHECK: 21
// TEST_FEATURE: Image_image_data_get_data_type

void foo(hipResourceType res21) {}

int main() {
  hipResourceDesc res21;
  foo(res21.resType);
  return 0;
}
