
#include <hip/hip_runtime.h>
// ====------ Image_api_test24.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test24_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test24_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test24_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test24_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test24_out

// CHECK: 27
// TEST_FEATURE: Image_sampling_info_set_coordinate_normalization_mode

int main() {
  hipTextureDesc tex_tmp;
  int normalized;
  tex_tmp.normalizedCoords = normalized;
  return 0;
}
