
#include <hip/hip_runtime.h>
// ====------ Image_api_test11.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test11_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test11_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test11_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test11_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test11_out

// CHECK: 27
// TEST_FEATURE: Image_image_wrapper_base_get_data

int main() {
  hipTextureObject_t tex42;
  hipResourceDesc res42;
  hipGetTextureObjectResourceDesc(&res42, tex42);
  return 0;
}
