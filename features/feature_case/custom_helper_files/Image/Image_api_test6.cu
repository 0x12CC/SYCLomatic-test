
#include <hip/hip_runtime.h>
// ====------ Image_api_test6.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test6_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test6_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test6_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test6_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test6_out

// CHECK: 20
// TEST_FEATURE: Image_image_data

int main() {
  hipResourceDesc a;
  return 0;
}
