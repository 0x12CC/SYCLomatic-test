// ====------ BlasUtils_api_test3.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test3_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test3_out

// CHECK: 5

// TEST_FEATURE: BlasUtils_get_value


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  hipblasHandle_t handle;
  float alpha_S, *x_S, *result_S;
  int incx, incy;
  hipblasSaxpy(handle, 10, &alpha_S, x_S, incx, result_S, incy);
  return 0;
}
