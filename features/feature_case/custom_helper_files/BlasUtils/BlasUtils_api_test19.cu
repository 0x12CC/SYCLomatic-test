// ====------ BlasUtils_api_test19.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_rot

int main() {
  hipblasHandle_t handle;
  void *x;
  void *y;
  void *sin;
  void *cos;

  hipblasRotEx(handle, 4, x, HIP_R_32F, 1,  y, HIP_R_32F, 1,  cos, sin, HIP_R_32F, HIP_R_32F);
  return 0;
}
