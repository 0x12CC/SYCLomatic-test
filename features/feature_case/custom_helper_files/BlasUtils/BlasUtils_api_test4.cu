// ====------ BlasUtils_api_test4.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test4_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test4_out

// CHECK: 35


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_getrf_batch_wrapper

int main() {
  hipblasHandle_t handle;
  int n = 275;
  int lda = 275;

  float **Aarray_S = 0;
  int *PivotArray = 0;
  int *infoArray = 0;
  int batchSize = 10;

  hipblasSgetrfBatched(handle, n, Aarray_S, lda, PivotArray, infoArray, batchSize);

  return 0;
}
