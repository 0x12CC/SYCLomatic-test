// ====------ BlasUtils_api_test25.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_gemm_batch

int main() {
  hipblasHandle_t handle;
  void * alpha;
  void * beta;
  const void** a;
  const void** b;
  void** c;

  hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, b, HIP_R_16F, 4, beta, c, HIP_R_16F, 4, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  return 0;
}
