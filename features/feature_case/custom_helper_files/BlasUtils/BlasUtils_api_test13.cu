// ====------ BlasUtils_api_test13.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_dotc

int main() {
  hipblasHandle_t handle;
  const void *x;
  const void *y;
  void *res;

  hipblasDotcEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  return 0;
}
