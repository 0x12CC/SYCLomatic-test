// ====------ BlasUtils_api_test22.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_axpy

int main() {
  hipblasHandle_t handle;
  const void *alpha;
  const void *x;
  void *y;

  hipblasAxpyEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, y, HIP_R_32F, 1, HIP_R_32F);
  return 0;
}
