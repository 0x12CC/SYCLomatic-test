// ====------ BlasUtils_api_test31.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_scal

int main() {
  hipblasHandle_t handle;
  void * alpha;
  void * x;

  hipblasScalEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, HIP_R_32F);
  return 0;
}
