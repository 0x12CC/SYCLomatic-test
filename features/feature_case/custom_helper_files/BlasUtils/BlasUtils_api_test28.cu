// ====------ BlasUtils_api_test28.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_herk

int main() {
  hipblasHandle_t handle;

  float2 *alpha;
  float *beta;
  float2 *a;
  float2 *b;
  float2 *c;

  hipblasCherkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, 2, 3, alpha, a, 3, b, 3, beta, c, 2);
  return 0;
}
