// ====------ BlasUtils_api_test5.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/BlasUtils/api_test5_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test5_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test5_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test5_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test5_out

// CHECK: 5


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_getrf_batch_wrapper

int main() {
  hipblasHandle_t handle;
  int n = 275;
  int lda = 275;

  float **Aarray_S = 0;
  int *PivotArray = 0;
  int *infoArray = 0;
  int batchSize = 10;

  hipblasSgetrfBatched(handle, n, Aarray_S, lda, PivotArray, infoArray, batchSize);
  return 0;
}
