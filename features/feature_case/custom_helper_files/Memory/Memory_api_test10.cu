
#include <hip/hip_runtime.h>
// ====------ Memory_api_test10.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test10_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test10_out

// CHECK: 3
// TEST_FEATURE: Memory_pitched_data

int main() {
  hipPitchedPtr a;
  return 0;
}
