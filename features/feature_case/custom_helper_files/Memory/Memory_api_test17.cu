
#include <hip/hip_runtime.h>
// ====------ Memory_api_test17.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test17_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test17_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test17_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test17_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test17_out

// CHECK: 39
// TEST_FEATURE: Memory_constant_memory_alias

__constant__ float A[1024];

int main() {
  return 0;
}
