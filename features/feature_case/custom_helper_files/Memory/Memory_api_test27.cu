
#include <hip/hip_runtime.h>
// ====------ Memory_api_test27.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test27_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test27_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test27_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test27_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test27_out

// CHECK: 48
// TEST_FEATURE: Memory_device_memory_get_ptr

__constant__ float constData[4];

int main() {
  float* host;
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), host, 1, 3, hipMemcpyHostToDevice, 0);
  return 0;
}
