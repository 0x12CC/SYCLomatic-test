
#include <hip/hip_runtime.h>
// ====------ Memory_api_test19.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test19_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test19_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test19_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test19_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test19_out

// CHECK: 39
// TEST_FEATURE: Memory_shared_memory_alias

__managed__ float A[1024];

int main() {
  return 0;
}
