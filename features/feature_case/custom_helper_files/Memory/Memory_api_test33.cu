
#include <hip/hip_runtime.h>
// ====------ Memory_api_test33.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test33_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test33_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test33_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test33_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test33_out

// CHECK: 27
// TEST_FEATURE: Memory_get_access

__global__ void foo(float* f) {}

int main() {
  float* f;
  hipMalloc(&f, 8);
  foo<<<1, 1>>>(f);
  return 0;
}
