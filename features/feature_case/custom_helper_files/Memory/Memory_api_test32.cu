
#include <hip/hip_runtime.h>
// ====------ Memory_api_test32.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test32_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test32_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test32_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test32_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test32_out

// CHECK: 28
// TEST_FEATURE: Memory_access_wrapper

__global__ void foo(float* f) {}

int main() {
  float* f;
  hipMalloc(&f, 8);
  f = f + 1;
  foo<<<1, 1>>>(f);
  return 0;
}