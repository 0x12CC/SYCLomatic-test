
#include <hip/hip_runtime.h>
// ====------ Memory_api_test18.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test18_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test18_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test18_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test18_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test18_out

// CHECK: 45
// TEST_FEATURE: Memory_shared_memory_alias

__managed__ float A[1024];

int main() {
  return 0;
}
