
#include <hip/hip_runtime.h>
// ====------ Memory_api_test9.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test9_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test9_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test9_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test9_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test9_out

// CHECK: 2
// TEST_FEATURE: Memory_memcpy_direction

void foo(hipMemcpyKind a) {}

int main() {
  return 0;
}
