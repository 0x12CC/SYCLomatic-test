
#include <hip/hip_runtime.h>
// ====------ Memory_api_test37.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test37_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test37_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test37_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test37_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test37_out

// CHECK: 29
// TEST_FEATURE: Memory_dpct_memcpy
// TEST_FEATURE: Memory_dpct_memcpy_2d
// TEST_FEATURE: Memory_dpct_memcpy_3d

int main() {
  hipMemcpy3DParms parms;
  hipMemcpy3D(&parms);
  return 0;
}
