
#include <hip/hip_runtime.h>
// ====------ Memory_api_test7.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test7_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test7_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test7_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test7_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test7_out

// CHECK: 16
// TEST_FEATURE: Memory_dpct_free

int main() {
  float* a;
  hipFree(a);
  return 0;
}
