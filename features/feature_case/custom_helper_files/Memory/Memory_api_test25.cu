
#include <hip/hip_runtime.h>
// ====------ Memory_api_test25.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test25_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test25_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test25_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test25_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test25_out

// CHECK: 53
// TEST_FEATURE: Memory_device_memory_get_access
// TEST_FEATURE: Memory_device_memory_init_q
// TEST_FEATURE: Memory_dpct_accessor

__device__ float c[16][16];

__global__ void kernel() {
  c[0][0] = 1.0f;
}

int main() {
  hipStream_t s;
  kernel<<<1, 1, 0, s>>>();
  return 0;
}
