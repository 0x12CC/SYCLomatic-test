
#include <hip/hip_runtime.h>
// ====------ Memory_api_test38.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test38_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test38_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test38_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test38_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test38_out

// CHECK: 37
// TEST_FEATURE: Memory_async_dpct_memcpy
// TEST_FEATURE: Memory_async_dpct_memcpy_2d
// TEST_FEATURE: Memory_async_dpct_memcpy_3d

int main() {
  hipMemcpy3DParms parms;
  hipMemcpy3DAsync(&parms);
  return 0;
}
