
#include <hip/hip_runtime.h>
// ====------ Memory_api_test24.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test24_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test24_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test24_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test24_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test24_out

// CHECK: 42
// TEST_FEATURE: Memory_device_memory_get_access
// TEST_FEATURE: Memory_device_memory_init
// TEST_FEATURE: Memory_dpct_accessor

__device__ float c[16][16];

__global__ void kernel() {
  c[0][0] = 1.0f;
}

int main() {
  kernel<<<1, 1>>>();
  return 0;
}
