
#include <hip/hip_runtime.h>
// ====------ Memory_api_test35.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test35_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test35_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test35_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test35_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test35_out

// CHECK: 27
// TEST_FEATURE: Memory_dpct_memcpy

int main() {
  float constData[1234567 * 4];
  float* h_A;
  int size;
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  return 0;
}
