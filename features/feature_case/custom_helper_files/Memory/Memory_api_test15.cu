
#include <hip/hip_runtime.h>
// ====------ Memory_api_test15.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test15_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test15_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test15_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test15_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test15_out

// CHECK: 39
// TEST_FEATURE: Memory_global_memory_alias

__device__ float A[1024];

int main() {
  return 0;
}
