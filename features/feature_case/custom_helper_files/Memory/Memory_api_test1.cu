
#include <hip/hip_runtime.h>
// ====------ Memory_api_test1.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test1_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test1_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test1_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test1_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test1_out

// CHECK: 27
// TEST_FEATURE: Memory_async_dpct_memset
// TEST_FEATURE: Memory_async_dpct_memset_2d
// TEST_FEATURE: Memory_async_dpct_memset_3d

int main() {
  hipExtent e = make_hipExtent(1, 1, 1);
  hipPitchedPtr p_A;
  hipMemset3DAsync(p_A, 0xf, e);
  return 0;
}
