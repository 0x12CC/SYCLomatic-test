
#include <hip/hip_runtime.h>
// ====------ Memory_api_test21.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test21_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test21_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test21_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test21_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test21_out

// CHECK: 40
// TEST_FEATURE: Memory_device_memory_get_size

static __device__ float d_A[1234567];

int main() {
  size_t size2;
  hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));
  return 0;
}
