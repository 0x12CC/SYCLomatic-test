
#include <hip/hip_runtime.h>
// ====------ Memory_api_test3.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test3_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test3_out

// CHECK: 24
// TEST_FEATURE: Memory_dpct_malloc_3d
// TEST_FEATURE: Memory_dpct_malloc_2d

int main() {
  hipExtent extent = make_hipExtent(1, 1, 1);
  hipPitchedPtr p3;
  hipMalloc3D(&p3, extent);
  return 0;
}
