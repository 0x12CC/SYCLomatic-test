
#include <hip/hip_runtime.h>
// ====------ Memory_api_test6.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test6_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test6_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test6_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test6_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test6_out

// CHECK: 24
// TEST_FEATURE: Memory_dpct_free

int main() {
  float* a;
  hipFree(a);
  return 0;
}
