
#include <hip/hip_runtime.h>
// ====------ Memory_api_test28.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test28_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test28_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test28_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test28_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test28_out

// CHECK: 41
// TEST_FEATURE: Memory_device_memory_get_ptr

__constant__ float constData[4];

int main() {
  float* host;
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), host, 1, 3, hipMemcpyHostToDevice, 0);
  return 0;
}
