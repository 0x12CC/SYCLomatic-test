// ====------ LibCommonUtils_api_test2.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// TEST_FEATURE: LibCommonUtils_mkl_get_version


#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

int main() {
  int ver;
  hipfftGetVersion(&ver);
  return 0;
}

