// ====------ RngUtils_api_test1.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// CHECK: 5
// TEST_FEATURE: RngUtils_rng_generator
// TEST_FEATURE: RngUtils_rng_generator_generate


#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
__device__ void foo() {
  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(1, 2, 3, &rng);
  hiprand(&rng);
}

int main() {
  return 0;
}
