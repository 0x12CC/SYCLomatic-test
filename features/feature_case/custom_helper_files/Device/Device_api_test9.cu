// ====------ Device_api_test9.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/Device/api_test9_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test9_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test9_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test9_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test9_out

// CHECK: 18
// TEST_FEATURE: Device_device_ext_get_saved_queue


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  float* x_S;
  int res = cublasIsamax(10, x_S, 1);
  return 0;
}
