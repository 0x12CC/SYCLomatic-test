
#include <hip/hip_runtime.h>
// ====------ Device_api_test1.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test1_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test1_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test1_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test1_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test1_out

// CHECK: 15
// TEST_FEATURE: Device_dev_mgr_get_device
// TEST_FEATURE: Device_device_ext_get_major_version
// TEST_FEATURE: Device_device_ext_get_minor_version

int main() {
  int result1, result2;
  hipDevice_t device;
  hipDeviceComputeCapability(&result1, &result2, device);
  return 0;
}
