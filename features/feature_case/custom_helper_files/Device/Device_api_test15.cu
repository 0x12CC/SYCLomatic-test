
#include <hip/hip_runtime.h>
// ====------ Device_api_test15.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test15_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test15_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test15_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test15_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test15_out

// CHECK: 11
// TEST_FEATURE: Device_dev_mgr_device_count

int main() {
  int result1;
  hipGetDeviceCount(&result1);
  return 0;
}
