
#include <hip/hip_runtime.h>
// ====------ Device_api_test13.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test13_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test13_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test13_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test13_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test13_out

// CHECK: 34
// TEST_FEATURE: Device_device_info_get_global_mem_size
// TEST_FEATURE: Device_get_current_device

int main() {
  size_t result1, result2;
  hipMemGetInfo(&result1, &result2);
  return 0;
}
