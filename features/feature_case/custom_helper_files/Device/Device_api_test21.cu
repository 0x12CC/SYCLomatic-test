
#include <hip/hip_runtime.h>
// ====------ Device_api_test21.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test21_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test21_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test21_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test21_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test21_out

// CHECK: 13
// TEST_FEATURE: Device_device_ext_is_native_atomic_supported

int main() {
  int res;
  hipDeviceGetAttribute(&res, hipDeviceAttributeHostNativeAtomicSupported, 0);
  return 0;
}
