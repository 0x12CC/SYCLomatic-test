
#include <hip/hip_runtime.h>
// ====------ Device_api_test18.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test18_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test18_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test18_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test18_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test18_out

// CHECK: 16
// TEST_FEATURE: Device_get_default_queue

__global__ void foo() {}

int main() {
  foo<<<1,1>>>();
  return 0;
}
