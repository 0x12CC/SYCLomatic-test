
#include <hip/hip_runtime.h>
// ====------ Device_api_test11.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test11_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test11_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test11_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test11_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test11_out

// CHECK: 16
// TEST_FEATURE: Device_device_ext_queues_wait_and_throw

int main() {
  hipDeviceSynchronize();
  return 0;
}
