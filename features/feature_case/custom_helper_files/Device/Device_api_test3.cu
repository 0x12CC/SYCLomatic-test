
#include <hip/hip_runtime.h>
// ====------ Device_api_test3.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test3_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test3_out

// CHECK: 16

// TEST_FEATURE: Device_device_ext_create_queue

int main() {
  hipStream_t s;
  hipStreamCreate(&s);
  return 0;
}
