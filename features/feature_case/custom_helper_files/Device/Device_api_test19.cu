
#include <hip/hip_runtime.h>
// ====------ Device_api_test19.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test19_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test19_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test19_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test19_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test19_out

// CHECK: 14
// TEST_FEATURE: Device_device_info_get_minor_version

int main() {
  int minor = 0;
  hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0);
  return 0;
}
