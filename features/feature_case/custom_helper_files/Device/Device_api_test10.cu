// ====------ Device_api_test10.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test10_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test10_out

// CHECK: 16
// TEST_FEATURE: Device_device_ext_set_saved_queue


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  hipStream_t stream;
  cublasSetKernelStream(stream);
  return 0;
}
