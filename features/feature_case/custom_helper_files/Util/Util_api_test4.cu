
#include <hip/hip_runtime.h>
// ====------ Util_api_test4.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test4_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test4_out

// CHECK: 2
// TEST_FEATURE: Util_vectorized_max

__device__ void foo() {
  unsigned u, u2;
  u = __vmaxs4(u, u2);
}

int main() {
  return 0;
}
