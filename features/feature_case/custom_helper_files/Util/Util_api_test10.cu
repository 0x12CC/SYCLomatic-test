// ====------ Util_api_test10.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test10_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test10_out

// CHECK: 29

// TEST_FEATURE: Util_matrix_mem_copy


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  float* a;
  hipblasSetVector(10, sizeof(float), a, 1, a, 1);
  return 0;
}
