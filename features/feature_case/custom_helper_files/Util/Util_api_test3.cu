
#include <hip/hip_runtime.h>
// ====------ Util_api_test3.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test3_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test3_out

// CHECK: 2
// TEST_FEATURE: Util_fast_length

__device__ void foo() {
  int i;
  double d;
  d = norm(i, &d);
}

int main() {
  return 0;
}
