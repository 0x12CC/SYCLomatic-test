
#include <hip/hip_runtime.h>
// ====------ Dpct_api_test3.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Dpct/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Dpct/api_test3_out/MainSourceFiles.yaml | wc -l > %T/Dpct/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/Dpct/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Dpct/api_test3_out

// CHECK: 2

// TEST_FEATURE: Dpct_dpct_compatibility_temp

#define AAA __CUDA_ARCH__

int main() {
  return 0;
}
