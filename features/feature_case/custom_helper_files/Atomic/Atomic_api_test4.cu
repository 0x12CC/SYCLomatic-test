
#include <hip/hip_runtime.h>
// ====------ Atomic_api_test4.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Atomic/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Atomic/api_test4_out/MainSourceFiles.yaml | wc -l > %T/Atomic/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/Atomic/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Atomic/api_test4_out

// CHECK: 2

// TEST_FEATURE: Atomic_atomic_fetch_and

__global__ void test(int *data) {
  int inc = 1;


  atomicAnd(&data[8], inc);

}
int main() {
  return 0;
}
