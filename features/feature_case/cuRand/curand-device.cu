// ====------ curand-device.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void my_kernel0() {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(1234, tid, 10, &rng);

  for (;;) {
    float2 aaa = hiprand_normal2(&rng);
    float2 bbb = hiprand_normal2(&rng);
  }
}

__global__ void my_kernel1(unsigned long seed, hiprandStateMRG32k3a_t *rngs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 10, &rngs[tid]);
}

__global__ void my_kernel2(double2 *res, hiprandStateMRG32k3a_t *rngs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  res[tid] = hiprand_normal2_double(&rngs[tid]);
}

int main() {
  my_kernel0<<<1, 1>>>();

  int size = 10;
  double2 *res;
  hiprandStateMRG32k3a_t *rngs;
  hipMalloc((void**)&rngs, size * sizeof(hiprandStateMRG32k3a_t));

  my_kernel1<<<1, 1>>>(1234, rngs);
  my_kernel2<<<1, 1>>>(res, rngs);

  return 0;
}

int foo() {
  int size = 10;
  hiprandStateMRG32k3a_t *rngs;
  hipMalloc((void**)&rngs, size * sizeof(hiprandStateMRG32k3a_t));
  my_kernel1<<<64, 128>>>(1234, rngs);
  return 0;
}

