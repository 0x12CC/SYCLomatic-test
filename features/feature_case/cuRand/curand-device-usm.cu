// ====------ curand-device-usm.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>

__global__ void my_kernel0() {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  hiprandState_t rng;
  hiprand_init(1234, tid, 0, &rng);

  for (;;) {
    float aaa = hiprand_uniform(&rng);
    float bbb = hiprand_uniform(&rng);
  }
}

__global__ void my_kernel1(unsigned long seed, hiprandState *rngs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &rngs[tid]);
}

__global__ void my_kernel2(double *res, hiprandState *rngs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  res[tid] = hiprand_normal_double(&rngs[tid]);
}

#define MY_CHECKER(c)                                                          \
{                                                                              \
    hipError_t error = c;                                                     \
    if (error != hipSuccess) { printf("error!\n"); }                          \
}

int main() {
  my_kernel0<<<1, 1>>>();

  int size = 10;
  double *res;
  hiprandState *rngs;
  void *rngs_temp;
  hipMalloc((void**)&rngs_temp, size * sizeof(hiprandState));
  rngs = (hiprandState*)rngs_temp;
  hipMalloc((void**)&rngs, size * sizeof(hiprandState) * 10);
  hipMalloc((void**)&rngs, size * sizeof(hiprandState));

  my_kernel1<<<1, 1>>>(1234, rngs);
  my_kernel2<<<1, 1>>>(res, rngs);

  int *dev_mem;
  MY_CHECKER(hipMalloc((void **)&dev_mem, sizeof(int) * 10));
  MY_CHECKER(hipMalloc((void **)&dev_mem, sizeof(hiprandState) * 10 * 10));
  dim3 grid(10, 1);
  MY_CHECKER(hipMalloc((void **)&dev_mem, sizeof(int) * grid.x));

  return 0;
}

