// ====------ curand-cross-function.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

void update(float* res, hiprandGenerator_t rng, long long aa, long long bb) {
  hiprandGenerateUniform(rng, res, aa * bb);
}

int main(){
  long long aa = 1024;
  long long bb = 1024;
  unsigned long long seed = 1234ULL;
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
  hiprandSetPseudoRandomGeneratorSeed(rng, seed);
  float *res;
  hiprandGenerateUniform(rng, res, aa * bb);
  update(res, rng, aa, bb);
  hiprandDestroyGenerator(rng);
}

