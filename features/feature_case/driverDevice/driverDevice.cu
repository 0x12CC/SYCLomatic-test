// ====------ driverDevice.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NUM 1
#define CUDA_SAFE_CALL( call) do {\
  int err = call;                \
} while (0)
int main(){
  int result1, result2;
  int *presult1 = &result1, *presult2 = &result2;
  hipDevice_t device;
  hipDevice_t *pdevice = &device;
  hipDeviceGet(&device, 0);
  hipDeviceGet(&device, NUM);
  hipDeviceGet(pdevice, 0);
  hipDeviceGet((hipDevice_t *)pdevice, 0);
  CUDA_SAFE_CALL(hipDeviceGet(&device, 0));
  hipDeviceGetAttribute(&result1, hipDeviceAttributeComputeCapabilityMajor, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeComputeCapabilityMinor, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeIntegrated, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeClockRate, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMultiprocessorCount, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeHostNativeAtomicSupported, device);
  CUDA_SAFE_CALL(hipDeviceGetAttribute(&result1, hipDeviceAttributeMultiprocessorCount, device));
  hipDeviceComputeCapability(&result1, &result2, device);
  CUDA_SAFE_CALL(hipDeviceComputeCapability(&result1, &result2, device));
  CUDA_SAFE_CALL(hipGetDeviceCount(&result1));
  hipGetDeviceCount(&result1);
  CUDA_SAFE_CALL(hipGetDeviceCount(&result1));

  char name[100];
  hipDeviceGetName(name, 90, device);
  CUDA_SAFE_CALL(hipDeviceGetName(name, 90, device));

  return 0;
}

