// ====------ cublas-only-usm.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int main(){
  hipblasHandle_t handle;
  const float** d_A_S_array;
  const float** d_B_S_array;
  float** d_C_S_array;
  const double** d_A_D_array;
  const double** d_B_D_array;
  double** d_C_D_array;
  const hipComplex** d_A_C_array = 0;
  const hipComplex** d_B_C_array = 0;
  hipComplex** d_C_C_array = 0;
  const hipDoubleComplex** d_A_Z_array = 0;
  const hipDoubleComplex** d_B_Z_array = 0;
  hipDoubleComplex** d_C_Z_array = 0;
  hipblasOperation_t trans3 = HIPBLAS_OP_N;
  int N = 10;
  float alpha_S;
  float beta_S;
  double alpha_D;
  double beta_D;
  hipComplex alpha_C;
  hipComplex beta_C;
  hipDoubleComplex alpha_Z;
  hipDoubleComplex beta_Z;

  int a = hipblasSgemmBatched(handle, trans3, trans3, N, N, N, &alpha_S, d_A_S_array, N, d_B_S_array, N, &beta_S, d_C_S_array, N, 10);
  hipblasDgemmBatched(handle, trans3, trans3, N, N, N, &alpha_D, d_A_D_array, N, d_B_D_array, N, &beta_D, d_C_D_array, N, 10);
  hipblasCgemmBatched(handle, trans3, trans3, N, N, N, &alpha_C, d_A_C_array, N, d_B_C_array, N, &beta_C, d_C_C_array, N, 10);
  hipblasZgemmBatched(handle, trans3, trans3, N, N, N, &alpha_Z, d_A_Z_array, N, d_B_Z_array, N, &beta_Z, d_C_Z_array, N, 10);

  a = hipblasStrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_S, d_A_S_array, N, d_C_S_array, N, 10);
  hipblasDtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_D, d_A_D_array, N, d_C_D_array, N, 10);
  hipblasCtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_C, d_A_C_array, N, d_C_C_array, N, 10);
  hipblasZtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_Z, d_A_Z_array, N, d_C_Z_array, N, 10);
}
