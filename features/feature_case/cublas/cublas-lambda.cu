// ====------ cublas-lambda.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

hipblasStatus_t status;
hipblasHandle_t handle;
int N = 275;
float *d_A_S = 0;
float *d_B_S = 0;
float *d_C_S = 0;
float alpha_S = 1.0f;
float beta_S = 0.0f;

int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  if (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)) {
  }
  else if (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)) {
  }


  if(int stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){
  }


  for(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);;){
  }

  for(;hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);){
  }

  while(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)!=0){
  }



  do{
  }while(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N));


  switch (int stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){
  }


  return 0;
}

int foo() {
  return hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
}

void foo2() {
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
}