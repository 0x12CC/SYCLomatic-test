// ====------ cublasReturnType.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

hipblasStatus_t foo(int m, int n) {
  return HIPBLAS_STATUS_SUCCESS;
}

hipblasHandle_t foo1(int m) {
  return 0;
}

hipComplex foo2(hipComplex m) {
  return make_hipComplex(1, 0);
}

hipDoubleComplex foo3(hipDoubleComplex m) {
  return make_hipDoubleComplex(1, 0);
}

hipblasOperation_t foo4(hipblasOperation_t m) {
  return HIPBLAS_OP_C;
}

hipblasFillMode_t foo5(hipblasFillMode_t m) {
  return HIPBLAS_FILL_MODE_LOWER;
}

hipblasSideMode_t foo6(hipblasSideMode_t m) {
  return HIPBLAS_SIDE_RIGHT;
}

hipblasDiagType_t foo7(hipblasDiagType_t m) {
  return HIPBLAS_DIAG_NON_UNIT;
}
