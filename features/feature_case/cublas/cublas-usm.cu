// ====------ cublas-usm.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

hipblasHandle_t handle;
int N = 275;
float *h_a, *h_b, *h_c;
const float *d_A_S;
const float *d_B_S;
float *d_C_S;
float alpha_S = 1.0f;
float beta_S = 0.0f;
int trans0 = 0;
int trans1 = 1;
int trans2 = 2;
int fill0 = 0;
int side0 = 0;
int diag0 = 0;
int *result = 0;
const float *x_S = 0;
const float *y_S = 0;

const double *d_A_D;
const double  *d_B_D;
double  *d_C_D;
double alpha_D;
double beta_D;
const double *x_D;
const double *y_D;

const float2 *d_A_C;
const float2  *d_B_C;
float2  *d_C_C;
float2 alpha_C;
float2 beta_C;
const float2 *x_C;
const float2 *y_C;

const double2 *d_A_Z;
const double2  *d_B_Z;
double2  *d_C_Z;
double2 alpha_Z;
double2 beta_Z;
const double2 *x_Z;
const double2 *y_Z;

float* result_S;
double* result_D;
float2* result_C;
double2* result_Z;

int incx, incy, lda, ldb, ldc;

int main() {

  int a = hipblasSetVector(10, sizeof(float), h_a, 11111, d_C_S, 11111);
  hipblasSetVector(10, sizeof(float), h_b, 1, d_C_S, 1);
  hipblasSetVector(10, sizeof(float), h_c, 1, d_C_S, 1);
  a = hipblasSetMatrix(100, 100, 10000, h_a, 100, d_C_S, 100);


  hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_DEVICE;
  hipblasGetPointerMode(handle, &mode);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

  //level 1

  a = hipblasIsamax(handle, N, x_S, N, result);
  hipblasIdamax(handle, N, x_D, N, result);
  a = hipblasIcamax(handle, N, x_C, N, result);
  hipblasIzamax(handle, N, x_Z, N, result);

  a = hipblasIsamin(handle, N, x_S, N, result);
  hipblasIdamin(handle, N, x_D, N, result);
  a = hipblasIcamin(handle, N, x_C, N, result);
  hipblasIzamin(handle, N, x_Z, N, result);

  a = hipblasSrotm(handle, N, d_C_S, N, d_C_S, N, x_S);
  hipblasDrotm(handle, N, d_C_D, N, d_C_D, N, x_D);

  a = hipblasScopy(handle, N, x_S, incx, d_C_S, incy);
  hipblasDcopy(handle, N, x_D, incx, d_C_D, incy);
  a = hipblasCcopy(handle, N, x_C, incx, d_C_C, incy);
  hipblasZcopy(handle, N, x_Z, incx, d_C_Z, incy);


  a = hipblasSaxpy(handle, N, &alpha_S, x_S, incx, result_S, incy);
  hipblasDaxpy(handle, N, &alpha_D, x_D, incx, result_D, incy);
  a = hipblasCaxpy(handle, N, &alpha_C, x_C, incx, result_C, incy);
  hipblasZaxpy(handle, N, &alpha_Z, x_Z, incx, result_Z, incy);

  a = hipblasSscal(handle, N, &alpha_S, result_S, incx);
  hipblasDscal(handle, N, &alpha_D, result_D, incx);
  a = hipblasCscal(handle, N, &alpha_C, result_C, incx);
  hipblasZscal(handle, N, &alpha_Z, result_Z, incx);

  a = hipblasSnrm2(handle, N, x_S, incx, result_S);
  hipblasDnrm2(handle, N, x_D, incx, result_D);
  a = hipblasScnrm2(handle, N, x_C, incx, result_S);
  hipblasDznrm2(handle, N, x_Z, incx, result_D);

  a = hipblasSasum(handle, N, x_S, incx, result_S);
  hipblasDasum(handle, N, x_D, incx, result_D);
  a = hipblasScasum(handle, N, x_C, incx, result_S);
  hipblasDzasum(handle, N, x_Z, incx, result_D);

  float *a_S, *b_S, *c_S, *s_S;
  double *a_D, *b_D, *c_D, *s_D;
  float2 *a_C, *b_C, *s_C;
  double2 *a_Z, *b_Z, *s_Z;

  a = hipblasSrotg(handle, a_S, b_S, c_S, s_S);
  hipblasDrotg(handle, a_D, b_D, c_D, s_D);
  a = hipblasCrotg(handle, a_C, b_C, c_S, s_C);
  hipblasZrotg(handle, a_Z, b_Z, c_D, s_Z);

  const float *y1_S;
  const double *y1_D;
  a = hipblasSrotmg(handle, a_S, b_S, c_S, y1_S, s_S);
  hipblasDrotmg(handle, a_D, b_D, c_D, y1_D, s_D);


  a = hipblasSdot(handle, N, x_S, incx, y_S, incy, result_S);
  hipblasDdot(handle, N, x_D, incx, y_D, incy, result_D);

  a = hipblasCdotc(handle, N, x_C, incx, y_C, incy, result_C);
  hipblasZdotc(handle, N, x_Z, incx, y_Z, incy, result_Z);

  a = hipblasCdotu(handle, N, x_C, incx, y_C, incy, result_C);
  hipblasZdotu(handle, N, x_Z, incx, y_Z, incy, result_Z);

  //level 2

  a = hipblasSgemv(handle, (hipblasOperation_t)trans2, N, N, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasDgemv(handle, HIPBLAS_OP_N, N, N, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  a = hipblasCgemv(handle, (hipblasOperation_t)trans2, N, N, &alpha_C, x_C, lda, y_C, incx, &beta_C, result_C, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, N, N, &alpha_Z, x_Z, lda, y_Z, incx, &beta_Z, result_Z, incy);

  a = hipblasSger(handle, N, N, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasDger(handle, N, N, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  a = hipblasCgeru(handle, N, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  hipblasCgerc(handle, N, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  a = hipblasZgeru(handle, N, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);
  hipblasZgerc(handle, N, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);








  //level 3

  a = hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  a = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);

  a = hipblasSgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, 16, d_B_S, N, 16, &beta_S, d_C_S, N, 16, 10);
  hipblasDgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_D, d_A_D, N, 16, d_B_D, N, 16, &beta_D, d_C_D, N, 16, 10);
  a = hipblasCgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, 16, d_B_C, N, 16, &beta_C, d_C_C, N, 16, 10);
  hipblasZgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, 16, d_B_Z, N, 16, &beta_Z, d_C_Z, N, 16, 10);

  __half *d_A_H = 0;
  __half *d_B_H = 0;
  __half *d_C_H = 0;
  __half alpha_H;
  __half beta_H;
  hipblasOperation_t trans3 = HIPBLAS_OP_N;
  a = hipblasHgemm(handle, trans3, trans3, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);

  void *alpha, *beta, *A, *B, *C;
  hipblasGemmAlgo_t algo = CUBLAS_GEMM_ALGO0;
  
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_32F, N, B, HIP_R_32F, N, beta, C, HIP_R_32F, N, HIP_R_32F, algo);

  float2 alpha_C, beta_C;
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_16F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_32F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_32F, N, B, HIP_R_32F, N, &beta_S, C, HIP_R_32F, N);
  cublasCgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_C, A, HIP_C_32F, N, B, HIP_C_32F, N, &beta_C, C, HIP_C_32F, N);

  const float** d_A_S_array;
  const float** d_B_S_array;
  float** d_C_S_array;
  const double** d_A_D_array;
  const double** d_B_D_array;
  double** d_C_D_array;
  const hipComplex** d_A_C_array = 0;
  const hipComplex** d_B_C_array = 0;
  hipComplex** d_C_C_array = 0;
  const hipDoubleComplex** d_A_Z_array = 0;
  const hipDoubleComplex** d_B_Z_array = 0;
  hipDoubleComplex** d_C_Z_array = 0;

  a = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N);
  hipblasDtrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_D, d_A_D, N, d_B_D, N, d_C_D, N);
  a = hipblasCtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, N, &alpha_C, d_A_C, N, d_B_C, N, d_C_C, N);
  hipblasZtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, d_C_Z, N);


  a = hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasDsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);



  if(int stat = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N)){}

  if(int stat = hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){}


}

int foo1(){
  return hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N);
}

int foo2(){
  return hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
}

void foo3() {
  hipblasHandle_t handle;
  float   *a_f, *b_f, *x_f, *c_f, *alpha_f, *beta_f;
  double  *a_d, *b_d, *x_d, *c_d, *alpha_d, *beta_d;
  float2  *a_c, *b_c, *x_c, *c_c, *alpha_c, *beta_c;
  double2 *a_z, *b_z, *x_z, *c_z, *alpha_z, *beta_z;

  hipblasSsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_f, a_f, 3, b_f, 3, beta_f, c_f, 2);
  hipblasDsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_d, a_d, 3, b_d, 3, beta_d, c_d, 2);
  hipblasCsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_c, c_c, 2);
  hipblasZsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_z, c_z, 2);
  hipblasCherkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_f, c_c, 2);
  hipblasZherkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_d, c_z, 2);

  hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_f, 2, x_f, 1, c_f, 2);
  hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_d, 2, x_d, 1, c_d, 2);
  hipblasCdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_c, 2, x_c, 1, c_c, 2);
  hipblasZdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_z, 2, x_z, 1, c_z, 2);
}
