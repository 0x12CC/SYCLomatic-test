// ====------ cublasLegacyHelper.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

#define MACRO_A cublasInit()

#define MACRO_B(status) (status)

#define MACRO_C(pointer) status = cublasFree(d_A)

void foo2(hipblasStatus_t){}

void foo(hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t) {}

void bar(hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t) {}

hipblasStatus_t foo(int m, int n) {
  return HIPBLAS_STATUS_SUCCESS;
}

int main() {
  foo(HIPBLAS_STATUS_SUCCESS, HIPBLAS_STATUS_NOT_INITIALIZED, HIPBLAS_STATUS_ALLOC_FAILED, HIPBLAS_STATUS_INVALID_VALUE, HIPBLAS_STATUS_ARCH_MISMATCH, HIPBLAS_STATUS_MAPPING_ERROR, HIPBLAS_STATUS_EXECUTION_FAILED, HIPBLAS_STATUS_INTERNAL_ERROR, HIPBLAS_STATUS_NOT_SUPPORTED, HIPBLAS_STATUS_UNKNOWN);
  bar(HIPBLAS_STATUS_SUCCESS, HIPBLAS_STATUS_NOT_INITIALIZED, HIPBLAS_STATUS_ALLOC_FAILED, HIPBLAS_STATUS_INVALID_VALUE, HIPBLAS_STATUS_ARCH_MISMATCH, HIPBLAS_STATUS_MAPPING_ERROR, HIPBLAS_STATUS_EXECUTION_FAILED, HIPBLAS_STATUS_INTERNAL_ERROR, HIPBLAS_STATUS_NOT_SUPPORTED, HIPBLAS_STATUS_UNKNOWN);

  hipblasStatus_t status;
  status = cublasInit();
  cublasInit();
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  status = MACRO_A;

  int a = sizeof(hipblasStatus_t);
  a = sizeof(hipblasStatus_t);
  a = sizeof(hipblasHandle_t);
  a = sizeof(hipComplex);
  a = sizeof(hipDoubleComplex);

  hipStream_t stream1;
  hipStreamCreate(&stream1);
  cublasSetKernelStream(stream1);
  cublasErrCheck(cublasSetKernelStream(stream1));

  float *d_A = NULL;
  int n = 10;
  int elemSize = 4;

  status = cublasAlloc(n, elemSize, (void **)&d_A);
  cublasAlloc(n, elemSize, (void **)&d_A);

  foo2(cublasAlloc(n, elemSize, (void **)&d_A));

  status = cublasFree(d_A);
  cublasFree(d_A);

  foo2(cublasFree(d_A));

  MACRO_B(cublasFree(d_A));

  MACRO_B(cublasGetError());

  MACRO_C(d_A);

  cublasGetError();
  status = cublasGetError();

  foo2(cublasGetError());

  foo2(cublasShutdown());

  foo2(cublasInit());

  status = cublasShutdown();
  cublasShutdown();
  return 0;
}
