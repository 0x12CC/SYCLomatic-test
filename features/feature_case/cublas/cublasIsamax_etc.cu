// ====------ cublasIsamax_etc.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int foo();

int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int k = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;
  const float *A_S = 0;
  const float *B_S = 0;
  float *C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  const double *A_D = 0;
  const double *B_D = 0;
  double *C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;

  const float *x_S = 0;
  const double *x_D = 0;
  const float *y_S = 0;
  const double *y_D = 0;
  int incx = 1;
  int incy = 1;
  int *result = 0;
  float *result_S = 0;
  double *result_D = 0;
  //level1
  //cublasI<t>amax
  status = hipblasIsamax(handle, n, x_S, incx, result);
  hipblasIsamax(handle, n, x_S, incx, result);

  status = hipblasIdamax(handle, n, x_D, incx, result);
  hipblasIdamax(handle, n, x_D, incx, result);

  //cublasI<t>amin
  status = hipblasIsamin(handle, n, x_S, incx, result);
  hipblasIsamin(handle, n, x_S, incx, result);

  status = hipblasIdamin(handle, n, x_D, incx, result);
  hipblasIdamin(handle, n, x_D, incx, result);

  //cublas<t>asum
  status = hipblasSasum(handle, n, x_S, incx, result_S);
  hipblasSasum(handle, n, x_S, incx, result_S);

  status = hipblasDasum(handle, n, x_D, incx, result_D);
  hipblasDasum(handle, n, x_D, incx, result_D);

  //cublas<t>axpy
  status = hipblasSaxpy(handle, n, &alpha_S, x_S, incx, result_S, incy);
  hipblasSaxpy(handle, n, &alpha_S, x_S, incx, result_S, incy);

  status = hipblasDaxpy(handle, n, &alpha_D, x_D, incx, result_D, incy);
  hipblasDaxpy(handle, n, &alpha_D, x_D, incx, result_D, incy);

  //cublas<t>copy
  status = hipblasScopy(handle, n, x_S, incx, result_S, incy);
  hipblasScopy(handle, n, x_S, incx, result_S, incy);

  status = hipblasDcopy(handle, n, x_D, incx, result_D, incy);
  hipblasDcopy(handle, n, x_D, incx, result_D, incy);

  //cublas<t>dot
  status = hipblasSdot(handle, n, x_S, incx, y_S, incy, result_S);
  hipblasSdot(handle, n, x_S, incx, y_S, incy, result_S);

  status = hipblasDdot(handle, n, x_D, incx, y_D, incy, result_D);
  hipblasDdot(handle, n, x_D, incx, y_D, incy, result_D);

  //cublas<t>nrm2
  status = hipblasSnrm2(handle, n, x_S, incx, result_S);
  hipblasSnrm2(handle, n, x_S, incx, result_S);

  status = hipblasDnrm2(handle, n, x_D, incx, result_D);
  hipblasDnrm2(handle, n, x_D, incx, result_D);

  float *x_f = 0;
  float *y_f = 0;
  double *x_d = 0;
  double *y_d = 0;
  //cublas<t>rot
  status = hipblasSrot(handle, n, x_f, incx, y_f, incy, x_S, y_S);
  hipblasSrot(handle, n, x_f, incx, y_f, incy, x_S, y_S);

  status = hipblasDrot(handle, n, x_d, incx, y_d, incy, x_D, y_D);
  hipblasDrot(handle, n, x_d, incx, y_d, incy, x_D, y_D);

  //cublas<t>rotg
  status = hipblasSrotg(handle, x_f, y_f, x_f, y_f);
  hipblasSrotg(handle, x_f, y_f, x_f, y_f);

  status = hipblasDrotg(handle, x_d, y_d, x_d, y_d);
  hipblasDrotg(handle, x_d, y_d, x_d, y_d);

  //cublas<t>rotm
  status = hipblasSrotm(handle, n, x_f, incx, y_f, incy, x_S);
  hipblasSrotm(handle, n, x_f, incx, y_f, incy, x_S);

  status = hipblasDrotm(handle, n, x_d, incx, y_d, incy, x_D);
  hipblasDrotm(handle, n, x_d, incx, y_d, incy, x_D);

  //cublas<t>rotmg
  status = hipblasSrotmg(handle, x_f, y_f, y_f, x_S, y_f);
  hipblasSrotmg(handle, x_f, y_f, y_f, x_S, y_f);

  status = hipblasDrotmg(handle, x_d, y_d, y_d, x_D, y_d);
  hipblasDrotmg(handle, x_d, y_d, y_d, x_D, y_d);

  //cublas<t>scal
  status = hipblasSscal(handle, n, &alpha_S, x_f, incx);
  hipblasSscal(handle, n, &alpha_S, x_f, incx);

  status = hipblasDscal(handle, n, &alpha_D, x_d, incx);
  hipblasDscal(handle, n, &alpha_D, x_d, incx);

  //cublas<t>swap
  status = hipblasSswap(handle, n, x_f, incx, y_f, incy);
  hipblasSswap(handle, n, x_f, incx, y_f, incy);

  status = hipblasDswap(handle, n, x_d, incx, y_d, incy);
  hipblasDswap(handle, n, x_d, incx, y_d, incy);

  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  //level2
  //cublas<t>gbmv
  status = hipblasSgbmv(handle, (hipblasOperation_t)trans0, m, n, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  status = hipblasDgbmv(handle, (hipblasOperation_t)trans1, m, n, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>gemv
  status = hipblasSgemv(handle, (hipblasOperation_t)trans2, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  status = hipblasDgemv(handle, (hipblasOperation_t)0, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>ger
  status = hipblasSger(handle, m, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasSger(handle, m, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);

  status = hipblasDger(handle, m, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDger(handle, m, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);

  int fill0 = 0;
  int fill1 = 1;
  //cublas<t>sbmv
  status = hipblasSsbmv(handle, (hipblasFillMode_t)fill0, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  status = hipblasDsbmv(handle, (hipblasFillMode_t)fill1, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>spmv
  status = hipblasSspmv(handle, (hipblasFillMode_t)0, n, &alpha_S, x_S, y_S, incx, &beta_S, result_S, incy);
  hipblasSspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, y_S, incx, &beta_S, result_S, incy);

  status = hipblasDspmv(handle, (hipblasFillMode_t)1, n, &alpha_D, x_D, y_D, incx, &beta_D, result_D, incy);
  hipblasDspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>spr
  status = hipblasSspr(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, result_S);
  hipblasSspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S);

  status = hipblasDspr(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, result_D);
  hipblasDspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D);

  //cublas<t>spr2
  status = hipblasSspr2(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, y_S, incy, result_S);
  hipblasSspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S);

  status = hipblasDspr2(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, y_D, incy, result_D);
  hipblasDspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D);

  //cublas<t>symv
  status = hipblasSsymv(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  status = hipblasDsymv(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>syr
  status = hipblasSsyr(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, result_S, lda);
  hipblasSsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S, lda);

  status = hipblasDsyr(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, result_D, lda);
  hipblasDsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D, lda);

  //cublas<t>syr2
  status = hipblasSsyr2(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasSsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);

  status = hipblasDsyr2(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);

  int diag0 = 0;
  int diag1 = 1;
  //cublas<t>tbmv
  status = hipblasStbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)1, (hipblasDiagType_t)diag0, n, n, x_S, lda, result_S, incy);
  hipblasStbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);

  status = hipblasDtbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)2, (hipblasDiagType_t)diag1, n, n, x_D, lda, result_D, incy);
  hipblasDtbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);

  //cublas<t>tbsv
  status = hipblasStbsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)0, n, n, x_S, lda, result_S, incy);
  hipblasStbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);

  status = hipblasDtbsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)1, n, n, x_D, lda, result_D, incy);
  hipblasDtbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);

  //cublas<t>tpmv
  status = hipblasStpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, result_S, incy);
  hipblasStpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);

  status = hipblasDtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, result_D, incy);
  hipblasDtpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);

  //cublas<t>tpsv
  status = hipblasStpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, result_S, incy);
  hipblasStpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);

  status = hipblasDtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, result_D, incy);
  hipblasDtpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);

  //cublas<t>trmv
  status = hipblasStrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, lda, result_S, incy);
  hipblasStrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);

  status = hipblasDtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, lda, result_D, incy);
  hipblasDtrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);

  //cublas<t>trsv
  status = hipblasStrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, lda, result_S, incy);
  hipblasStrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);

  status = hipblasDtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, lda, result_D, incy);
  hipblasDtrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);

  //level3
  int side0 = 0;
  int side1 = 1;
  // cublas<T>gemmStridedBatched
  status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, n, &alpha_S, A_S, n, 16, B_S, n, 16, &beta_S, C_S, n, 16, 10);
  hipblasSgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, n, n, n, &alpha_S, A_S, n, 16, B_S, n, 16, &beta_S, C_S, n, 16, 10);

  status = hipblasDgemmStridedBatched(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, n, n, n, &alpha_D, A_D, n, 16, B_D, n, 16, &beta_D, C_D, n, 16, 10);
  hipblasDgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, n, n, n, &alpha_D, A_D, n, 16, B_D, n, 16, &beta_D, C_D, n, 16, 10);

  const float** A_S_array;
  const float** B_S_array;
  float** C_S_array;
  const double** A_D_array;
  const double** B_D_array;
  double** C_D_array;
  hipblasOperation_t trans3 = HIPBLAS_OP_N;

  // cublas<T>symm
  status = hipblasSsymm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);
  hipblasSsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, m, n, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);

  status = hipblasDsymm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill0, m, n, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);
  hipblasDsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, m, n, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);

  // cublas<T>syrk
  status = hipblasSsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_S, A_S, lda, &beta_S, C_S, ldc);
  hipblasSsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, &beta_S, C_S, ldc);

  status = hipblasDsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_D, A_D, lda, &beta_D, C_D, ldc);
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, &beta_D, C_D, ldc);

  // cublas<T>syr2k
  status = hipblasSsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);
  hipblasSsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);

  status = hipblasDsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);
  hipblasDsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);

  // cublas<T>trsm
  status = hipblasStrsm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_S, A_S, lda, C_S, ldc);
  hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_S, A_S, lda, C_S, ldc);

  status = hipblasDtrsm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_D, A_D, lda, C_D, ldc);
  hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_D, A_D, lda, C_D, ldc);


  hipblasDtrsm(handle, (hipblasSideMode_t)foo(), (hipblasFillMode_t)foo(), (hipblasOperation_t)foo(), (hipblasDiagType_t)foo(), m, n, &alpha_D, A_D, lda, C_D, ldc);
  return 0;
}
