// ====------ cublas-create-Sgemm-destroy.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

extern hipblasHandle_t handle2;

int foo2(hipDataType DT) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_HOST;
  hipblasGetPointerMode(handle, &mode);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipDataType cdt;
  hipDataType cbdt;

  hipblasAtomicsMode_t Atomicsmode;
  hipblasGetAtomicsMode(handle, &Atomicsmode);
  hipblasSetAtomicsMode(handle, Atomicsmode);

  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipblasSetStream(handle, stream1);
  status = hipblasSetStream(handle, stream1);
  hipblasGetStream(handle, &stream1);
  status = hipblasGetStream(handle, &stream1);


  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);

  __half *d_A_H = 0;
  __half *d_B_H = 0;
  __half *d_C_H = 0;
  __half alpha_H;
  __half beta_H;
  status = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);
  hipblasHgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);

  void *alpha, *beta, *A, *B, *C;

  status = hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_16F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_32F, N, B, HIP_R_32F, N, beta, C, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_64F, N, B, HIP_R_64F, N, beta, C, HIP_R_64F, N, HIP_R_64F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_C_32F, N, B, HIP_C_32F, N, beta, C, HIP_C_32F, N, HIP_C_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_C_64F, N, B, HIP_C_64F, N, beta, C, HIP_C_64F, N, HIP_C_64F, CUBLAS_GEMM_ALGO0);

  float2 alpha_C, beta_C;
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_16F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_32F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_32F, N, B, HIP_R_32F, N, &beta_S, C, HIP_R_32F, N);
  cublasCgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_C, A, HIP_C_32F, N, B, HIP_C_32F, N, &beta_C, C, HIP_C_32F, N);

  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}

void foo4() {
  hipblasHandle_t handle;
  float   *a_f, *b_f, *x_f, *c_f, *alpha_f, *beta_f;
  double  *a_d, *b_d, *x_d, *c_d, *alpha_d, *beta_d;
  float2  *a_c, *b_c, *x_c, *c_c, *alpha_c, *beta_c;
  double2 *a_z, *b_z, *x_z, *c_z, *alpha_z, *beta_z;

  hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_f, 2, x_f, 1, c_f, 2);
  hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_d, 2, x_d, 1, c_d, 2);
  hipblasCdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_c, 2, x_c, 1, c_c, 2);
  hipblasZdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_z, 2, x_z, 1, c_z, 2);
}
