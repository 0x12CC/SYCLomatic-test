// ====------ cusolverDnLn_cuda10-1-part2.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    int m = 0;
    int n = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);
    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;

    //CHECK: /*
    //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = (*(&Lwork) = mkl::lapack::potri_scratchpad_size<float>(**cusolverH, uplo, n, lda), 0);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = (*(&Lwork) = mkl::lapack::potri_scratchpad_size<double>(**cusolverH, uplo, n, lda), 0);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = (*(&Lwork) = mkl::lapack::potri_scratchpad_size<std::complex<float>>(**cusolverH, uplo, n, lda), 0);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = (*(&Lwork) = mkl::lapack::potri_scratchpad_size<std::complex<double>>(**cusolverH, uplo, n, lda), 0);
    status = hipsolverDnSpotri_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotri_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotri_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotri_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    //CHECK: mkl::lapack::potri(**cusolverH, uplo, n, (float*)&A_f, lda, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: mkl::lapack::potri(**cusolverH, uplo, n, (double*)&A_d, lda, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: mkl::lapack::potri(**cusolverH, uplo, n, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: mkl::lapack::potri(**cusolverH, uplo, n, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
}
