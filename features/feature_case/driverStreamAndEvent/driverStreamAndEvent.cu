
#include <hip/hip_runtime.h>
// ====------ driverStreamAndEvent.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
int main(){
  hipFunction_t f;
  hipStream_t s;
  hipEvent_t e;

  cuFuncSetCacheConfig(f, hipFuncCachePreferNone);

  hipStreamCreateWithFlags(&s, hipStreamDefault);
  hipStreamSynchronize(s);

  hipEventCreateWithFlags(&e, hipEventDefault);
  hipStreamWaitEvent(s, e, 0);

  hipEventRecord(e, s);
  hipEventSynchronize(e);

  hipError_t r;
  r = hipEventQuery(e);

  hipEvent_t start, end;
  hipEventRecord(start, s);
  hipEventRecord(end, s);
  hipEventSynchronize(start);
  hipEventSynchronize(end);
  float result_time;
  hipEventElapsedTime(&result_time, start, end);

  int rr;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, f);

  return 0;
}
