// ====------ get_library_version.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <iostream>

#include "hipblas.h"

void foo1() {
  int version = 0;
  cublasGetVersion(&version);
  std::cout << "foo1" << std::endl;
  std::cout << version << std::endl;
}

#include "hipblas.h"

void foo2() {
  hipblasHandle_t handle;
  int version = 0;
  cublasGetVersion(handle, &version);
  std::cout << "foo2" << std::endl;
  std::cout << version << std::endl;
}

#include "hipfft/hipfft.h"

void foo3() {
  int version = 0;
  hipfftGetVersion(&version);
  libraryPropertyType major_t = MAJOR_VERSION;
  libraryPropertyType minor_t = MINOR_VERSION;
  libraryPropertyType patch_t = PATCH_LEVEL;
  int major = 0, minor = 0, patch = 0;
  hipfftGetProperty(major_t, &major);
  hipfftGetProperty(minor_t, &minor);
  hipfftGetProperty(patch_t, &patch);
  std::cout << "foo3" << std::endl;
  std::cout << version << std::endl;
  std::cout << major << std::endl;
  std::cout << minor << std::endl;
  std::cout << patch << std::endl;
}

int main() {
  foo1();
  foo2();
  foo3();
  return 0;
}
