// ====------ cufft-type.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

size_t size;

int main() {
  hipfftReal fftreal;
  hipfftDoubleReal fftdreal;
  hipfftComplex fftcomplex;
  hipfftDoubleComplex fftdcomplex;
  hipComplex ccomplex;
  hipDoubleComplex cdcomplex;
  size = sizeof(hipfftReal);
  size = sizeof(hipfftDoubleReal);
  size = sizeof(hipfftComplex);
  size = sizeof(hipfftDoubleComplex);
  size = sizeof(hipComplex);
  size = sizeof(hipDoubleComplex);

  int forward = HIPFFT_FORWARD;
  int inverse = HIPFFT_BACKWARD;

  hipfftType_t fftt_t;
  hipfftType fftt;
  size = sizeof(hipfftType_t);
  size = sizeof(hipfftType);
  fftt = HIPFFT_R2C;
  fftt = HIPFFT_C2R;
  fftt = HIPFFT_C2C;
  fftt = HIPFFT_D2Z;
  fftt = HIPFFT_Z2D;
  fftt = HIPFFT_Z2Z;

  hipfftResult_t fftr_t;
  hipfftResult fftr;
  size = sizeof(hipfftResult_t);
  size = sizeof(hipfftResult);
  fftr = HIPFFT_SUCCESS;
  fftr = HIPFFT_INVALID_PLAN;
  fftr = HIPFFT_ALLOC_FAILED;
  fftr = HIPFFT_INVALID_TYPE;
  fftr = HIPFFT_INVALID_VALUE;
  fftr = HIPFFT_INTERNAL_ERROR;
  fftr = HIPFFT_EXEC_FAILED;
  fftr = HIPFFT_SETUP_FAILED;
  fftr = HIPFFT_INVALID_SIZE;
  fftr = HIPFFT_UNALIGNED_DATA;
  fftr = HIPFFT_INCOMPLETE_PARAMETER_LIST;
  fftr = HIPFFT_INVALID_DEVICE;
  fftr = HIPFFT_PARSE_ERROR;
  fftr = HIPFFT_NO_WORKSPACE;
  fftr = HIPFFT_NOT_IMPLEMENTED;
  fftr = CUFFT_LICENSE_ERROR;
  fftr = HIPFFT_NOT_SUPPORTED;

  return 0;
}


template<
typename A = hipfftReal,
typename B = hipfftDoubleReal,
typename C = hipfftComplex,
typename D = hipfftDoubleComplex,
typename E = hipComplex,
typename F = hipDoubleComplex,
typename G = hipfftType_t,
typename H = hipfftType,
typename J = hipfftResult_t,
typename K = hipfftResult>
void foo1(
hipfftReal a,
hipfftDoubleReal b,
hipfftComplex c,
hipfftDoubleComplex d,
hipComplex e,
hipDoubleComplex f,
hipfftType_t g,
hipfftType h,
hipfftResult_t j,
hipfftResult k
){}


template<
hipfftType A1 = HIPFFT_R2C,
hipfftType A2 = HIPFFT_C2R,
hipfftType A3 = HIPFFT_C2C,
hipfftType A4 = HIPFFT_D2Z,
hipfftType A5 = HIPFFT_Z2D,
hipfftType A6 = HIPFFT_Z2Z,
hipfftResult B1 = HIPFFT_SUCCESS,
hipfftResult B2 = HIPFFT_INVALID_PLAN,
hipfftResult B3 = HIPFFT_ALLOC_FAILED,
hipfftResult B4 = HIPFFT_INVALID_TYPE,
hipfftResult B5 = HIPFFT_INVALID_VALUE,
hipfftResult B6 = HIPFFT_INTERNAL_ERROR,
hipfftResult B7 = HIPFFT_EXEC_FAILED,
hipfftResult B8 = HIPFFT_SETUP_FAILED,
hipfftResult B9 = HIPFFT_INVALID_SIZE,
hipfftResult B10 = HIPFFT_UNALIGNED_DATA,
hipfftResult B11 = HIPFFT_INCOMPLETE_PARAMETER_LIST,
hipfftResult B12 = HIPFFT_INVALID_DEVICE,
hipfftResult B13 = HIPFFT_PARSE_ERROR,
hipfftResult B14 = HIPFFT_NO_WORKSPACE,
hipfftResult B15 = HIPFFT_NOT_IMPLEMENTED,
hipfftResult B16 = CUFFT_LICENSE_ERROR,
hipfftResult B17 = HIPFFT_NOT_SUPPORTED>
void foo2(
hipfftType a1 = HIPFFT_R2C,
hipfftType a2 = HIPFFT_C2R,
hipfftType a3 = HIPFFT_C2C,
hipfftType a4 = HIPFFT_D2Z,
hipfftType a5 = HIPFFT_Z2D,
hipfftType a6 = HIPFFT_Z2Z,
hipfftResult b1 = HIPFFT_SUCCESS,
hipfftResult b2 = HIPFFT_INVALID_PLAN,
hipfftResult b3 = HIPFFT_ALLOC_FAILED,
hipfftResult b4 = HIPFFT_INVALID_TYPE,
hipfftResult b5 = HIPFFT_INVALID_VALUE,
hipfftResult b6 = HIPFFT_INTERNAL_ERROR,
hipfftResult b7 = HIPFFT_EXEC_FAILED,
hipfftResult b8 = HIPFFT_SETUP_FAILED,
hipfftResult b9 = HIPFFT_INVALID_SIZE,
hipfftResult b10 = HIPFFT_UNALIGNED_DATA,
hipfftResult b11 = HIPFFT_INCOMPLETE_PARAMETER_LIST,
hipfftResult b12 = HIPFFT_INVALID_DEVICE,
hipfftResult b13 = HIPFFT_PARSE_ERROR,
hipfftResult b14 = HIPFFT_NO_WORKSPACE,
hipfftResult b15 = HIPFFT_NOT_IMPLEMENTED,
hipfftResult b16 = CUFFT_LICENSE_ERROR,
hipfftResult b17 = HIPFFT_NOT_SUPPORTED
){}


template<typename T>
hipfftReal foo3(){}

template<typename T>
hipfftDoubleReal foo4(){}

template<typename T>
hipfftComplex foo5(){}

template<typename T>
hipfftDoubleComplex foo6(){}

template<typename T>
hipComplex foo7(){}

template<typename T>
hipDoubleComplex foo8(){}

template<typename T>
hipfftType_t foo9(){}

template<typename T>
hipfftType foo10(){}


template<typename T>
hipfftResult_t foo12(){}

template<typename T>
hipfftResult foo13(){}
