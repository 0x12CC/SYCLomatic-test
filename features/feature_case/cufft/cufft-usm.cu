// ====------ cufft-usm.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


int main() {
  hipfftHandle plan_1d_C2C;
  float2* odata_1d_C2C;
  float2* idata_1d_C2C;

  hipfftPlan1d(&plan_1d_C2C, 10, HIPFFT_C2C, 3);

  hipfftExecC2C(plan_1d_C2C, idata_1d_C2C, odata_1d_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_1d_C2R;
  float* odata_1d_C2R;
  float2* idata_1d_C2R;

  hipfftPlan1d(&plan_1d_C2R, 10, HIPFFT_C2R, 3);

  hipfftExecC2R(plan_1d_C2R, idata_1d_C2R, odata_1d_C2R);

  hipfftHandle plan_1d_R2C;
  float2* odata_1d_R2C;
  float* idata_1d_R2C;

  hipfftPlan1d(&plan_1d_R2C, 10, HIPFFT_R2C, 3);

  hipfftExecR2C(plan_1d_R2C, idata_1d_R2C, odata_1d_R2C);

  hipfftHandle plan_1d_Z2Z;
  double2* odata_1d_Z2Z;
  double2* idata_1d_Z2Z;

  hipfftPlan1d(&plan_1d_Z2Z, 10, HIPFFT_Z2Z, 3);

  hipfftExecZ2Z(plan_1d_Z2Z, idata_1d_Z2Z, odata_1d_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_1d_Z2D;
  double* odata_1d_Z2D;
  double2* idata_1d_Z2D;

  hipfftPlan1d(&plan_1d_Z2D, 10, HIPFFT_Z2D, 3);

  hipfftExecZ2D(plan_1d_Z2D, idata_1d_Z2D, odata_1d_Z2D);

  hipfftHandle plan_1d_D2Z;
  double2* odata_1d_D2Z;
  double* idata_1d_D2Z;

  hipfftPlan1d(&plan_1d_D2Z, 10, HIPFFT_D2Z, 3);

  hipfftExecD2Z(plan_1d_D2Z, idata_1d_D2Z, odata_1d_D2Z);

  hipfftHandle plan_2d_C2C;
  float2* odata_2d_C2C;
  float2* idata_2d_C2C;

  hipfftPlan2d(&plan_2d_C2C, 10, 20, HIPFFT_C2C);

  hipfftExecC2C(plan_2d_C2C, idata_2d_C2C, odata_2d_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_2d_C2R;
  float* odata_2d_C2R;
  float2* idata_2d_C2R;

  hipfftPlan2d(&plan_2d_C2R, 10, 20, HIPFFT_C2R);

  hipfftExecC2R(plan_2d_C2R, idata_2d_C2R, odata_2d_C2R);

  hipfftHandle plan_2d_R2C;
  float2* odata_2d_R2C;
  float* idata_2d_R2C;

  hipfftPlan2d(&plan_2d_R2C, 10, 20, HIPFFT_R2C);

  hipfftExecR2C(plan_2d_R2C, idata_2d_R2C, odata_2d_R2C);

  hipfftHandle plan_2d_Z2Z;
  double2* odata_2d_Z2Z;
  double2* idata_2d_Z2Z;

  hipfftPlan2d(&plan_2d_Z2Z, 10, 20, HIPFFT_Z2Z);

  hipfftExecZ2Z(plan_2d_Z2Z, idata_2d_Z2Z, odata_2d_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_2d_Z2D;
  double* odata_2d_Z2D;
  double2* idata_2d_Z2D;

  hipfftPlan2d(&plan_2d_Z2D, 10, 20, HIPFFT_Z2D);

  hipfftExecZ2D(plan_2d_Z2D, idata_2d_Z2D, odata_2d_Z2D);

  hipfftHandle plan_2d_D2Z;
  double2* odata_2d_D2Z;
  double* idata_2d_D2Z;

  hipfftPlan2d(&plan_2d_D2Z, 10, 20, HIPFFT_D2Z);

  hipfftExecD2Z(plan_2d_D2Z, idata_2d_D2Z, odata_2d_D2Z);

  hipfftHandle plan_3d_C2C;
  float2* odata_3d_C2C;
  float2* idata_3d_C2C;

  hipfftPlan3d(&plan_3d_C2C, 10, 20, 30, HIPFFT_C2C);

  hipfftExecC2C(plan_3d_C2C, idata_3d_C2C, odata_3d_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_3d_C2R;
  float* odata_3d_C2R;
  float2* idata_3d_C2R;

  hipfftPlan3d(&plan_3d_C2R, 10, 20, 30, HIPFFT_C2R);

  hipfftExecC2R(plan_3d_C2R, idata_3d_C2R, odata_3d_C2R);

  hipfftHandle plan_3d_R2C;
  float2* odata_3d_R2C;
  float* idata_3d_R2C;

  hipfftPlan3d(&plan_3d_R2C, 10, 20, 30, HIPFFT_R2C);

  hipfftExecR2C(plan_3d_R2C, idata_3d_R2C, odata_3d_R2C);

  hipfftHandle plan_3d_Z2Z;
  double2* odata_3d_Z2Z;
  double2* idata_3d_Z2Z;

  hipfftPlan3d(&plan_3d_Z2Z, 10, 20, 30, HIPFFT_Z2Z);

  hipfftExecZ2Z(plan_3d_Z2Z, idata_3d_Z2Z, odata_3d_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_3d_Z2D;
  double* odata_3d_Z2D;
  double2* idata_3d_Z2D;

  hipfftPlan3d(&plan_3d_Z2D, 10, 20, 30, HIPFFT_Z2D);

  hipfftExecZ2D(plan_3d_Z2D, idata_3d_Z2D, odata_3d_Z2D);

  hipfftHandle plan_3d_D2Z;
  double2* odata_3d_D2Z;
  double* idata_3d_D2Z;

  hipfftPlan3d(&plan_3d_D2Z, 10, 20, 30, HIPFFT_D2Z);

  hipfftExecD2Z(plan_3d_D2Z, idata_3d_D2Z, odata_3d_D2Z);

  hipfftHandle plan_many_C2C;
  int odist_many_C2C;
  int ostride_many_C2C;
  int * onembed_many_C2C;
  int idist_many_C2C;
  int istride_many_C2C;
  int* inembed_many_C2C;
  int * n_many_C2C;
  float2* odata_many_C2C;
  float2* idata_many_C2C;

  hipfftPlanMany(&plan_many_C2C, 3, n_many_C2C, inembed_many_C2C, istride_many_C2C, idist_many_C2C, onembed_many_C2C, ostride_many_C2C, odist_many_C2C, HIPFFT_C2C, 12);

  hipfftExecC2C(plan_many_C2C, idata_many_C2C, odata_many_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_many_C2R;
  int odist_many_C2R;
  int ostride_many_C2R;
  int * onembed_many_C2R;
  int idist_many_C2R;
  int istride_many_C2R;
  int* inembed_many_C2R;
  int * n_many_C2R;
  float* odata_many_C2R;
  float2* idata_many_C2R;

  hipfftPlanMany(&plan_many_C2R, 3, n_many_C2R, inembed_many_C2R, istride_many_C2R, idist_many_C2R, onembed_many_C2R, ostride_many_C2R, odist_many_C2R, HIPFFT_C2R, 12);

  hipfftExecC2R(plan_many_C2R, idata_many_C2R, odata_many_C2R);

  hipfftHandle plan_many_R2C;
  int odist_many_R2C;
  int ostride_many_R2C;
  int * onembed_many_R2C;
  int idist_many_R2C;
  int istride_many_R2C;
  int* inembed_many_R2C;
  int * n_many_R2C;
  float2* odata_many_R2C;
  float* idata_many_R2C;

  hipfftPlanMany(&plan_many_R2C, 3, n_many_R2C, inembed_many_R2C, istride_many_R2C, idist_many_R2C, onembed_many_R2C, ostride_many_R2C, odist_many_R2C, HIPFFT_R2C, 12);

  hipfftExecR2C(plan_many_R2C, idata_many_R2C, odata_many_R2C);

  hipfftHandle plan_many_Z2Z;
  int odist_many_Z2Z;
  int ostride_many_Z2Z;
  int * onembed_many_Z2Z;
  int idist_many_Z2Z;
  int istride_many_Z2Z;
  int* inembed_many_Z2Z;
  int * n_many_Z2Z;
  double2* odata_many_Z2Z;
  double2* idata_many_Z2Z;

  hipfftPlanMany(&plan_many_Z2Z, 3, n_many_Z2Z, inembed_many_Z2Z, istride_many_Z2Z, idist_many_Z2Z, onembed_many_Z2Z, ostride_many_Z2Z, odist_many_Z2Z, HIPFFT_Z2Z, 12);

  hipfftExecZ2Z(plan_many_Z2Z, idata_many_Z2Z, odata_many_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_many_Z2D;
  int odist_many_Z2D;
  int ostride_many_Z2D;
  int * onembed_many_Z2D;
  int idist_many_Z2D;
  int istride_many_Z2D;
  int* inembed_many_Z2D;
  int * n_many_Z2D;
  double* odata_many_Z2D;
  double2* idata_many_Z2D;

  hipfftPlanMany(&plan_many_Z2D, 3, n_many_Z2D, inembed_many_Z2D, istride_many_Z2D, idist_many_Z2D, onembed_many_Z2D, ostride_many_Z2D, odist_many_Z2D, HIPFFT_Z2D, 12);

  hipfftExecZ2D(plan_many_Z2D, idata_many_Z2D, odata_many_Z2D);

  hipfftHandle plan_many_D2Z;
  int odist_many_D2Z;
  int ostride_many_D2Z;
  int * onembed_many_D2Z;
  int idist_many_D2Z;
  int istride_many_D2Z;
  int* inembed_many_D2Z;
  int * n_many_D2Z;
  double2* odata_many_D2Z;
  double* idata_many_D2Z;

  hipfftPlanMany(&plan_many_D2Z, 3, n_many_D2Z, inembed_many_D2Z, istride_many_D2Z, idist_many_D2Z, onembed_many_D2Z, ostride_many_D2Z, odist_many_D2Z, HIPFFT_D2Z, 12);

  hipfftExecD2Z(plan_many_D2Z, idata_many_D2Z, odata_many_D2Z);

  hipfftHandle plan_m1d_C2C;
  size_t* work_size_m1d_C2C;
  float2* odata_m1d_C2C;
  float2* idata_m1d_C2C;

  hipfftMakePlan1d(plan_m1d_C2C, 10, HIPFFT_C2C, 3, work_size_m1d_C2C);

  hipfftExecC2C(plan_m1d_C2C, idata_m1d_C2C, odata_m1d_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_m1d_C2R;
  size_t* work_size_m1d_C2R;
  float* odata_m1d_C2R;
  float2* idata_m1d_C2R;

  hipfftMakePlan1d(plan_m1d_C2R, 10, HIPFFT_C2R, 3, work_size_m1d_C2R);

  hipfftExecC2R(plan_m1d_C2R, idata_m1d_C2R, odata_m1d_C2R);

  hipfftHandle plan_m1d_R2C;
  size_t* work_size_m1d_R2C;
  float2* odata_m1d_R2C;
  float* idata_m1d_R2C;

  hipfftMakePlan1d(plan_m1d_R2C, 10, HIPFFT_R2C, 3, work_size_m1d_R2C);

  hipfftExecR2C(plan_m1d_R2C, idata_m1d_R2C, odata_m1d_R2C);

  hipfftHandle plan_m1d_Z2Z;
  size_t* work_size_m1d_Z2Z;
  double2* odata_m1d_Z2Z;
  double2* idata_m1d_Z2Z;

  hipfftMakePlan1d(plan_m1d_Z2Z, 10, HIPFFT_Z2Z, 3, work_size_m1d_Z2Z);

  hipfftExecZ2Z(plan_m1d_Z2Z, idata_m1d_Z2Z, odata_m1d_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_m1d_Z2D;
  size_t* work_size_m1d_Z2D;
  double* odata_m1d_Z2D;
  double2* idata_m1d_Z2D;

  hipfftMakePlan1d(plan_m1d_Z2D, 10, HIPFFT_Z2D, 3, work_size_m1d_Z2D);

  hipfftExecZ2D(plan_m1d_Z2D, idata_m1d_Z2D, odata_m1d_Z2D);

  hipfftHandle plan_m1d_D2Z;
  size_t* work_size_m1d_D2Z;
  double2* odata_m1d_D2Z;
  double* idata_m1d_D2Z;

  hipfftMakePlan1d(plan_m1d_D2Z, 10, HIPFFT_D2Z, 3, work_size_m1d_D2Z);

  hipfftExecD2Z(plan_m1d_D2Z, idata_m1d_D2Z, odata_m1d_D2Z);

  hipfftHandle plan_m2d_C2C;
  size_t* work_size_m2d_C2C;
  float2* odata_m2d_C2C;
  float2* idata_m2d_C2C;

  hipfftMakePlan2d(plan_m2d_C2C, 10, 20, HIPFFT_C2C, work_size_m2d_C2C);

  hipfftExecC2C(plan_m2d_C2C, idata_m2d_C2C, odata_m2d_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_m2d_C2R;
  size_t* work_size_m2d_C2R;
  float* odata_m2d_C2R;
  float2* idata_m2d_C2R;

  hipfftMakePlan2d(plan_m2d_C2R, 10, 20, HIPFFT_C2R, work_size_m2d_C2R);

  hipfftExecC2R(plan_m2d_C2R, idata_m2d_C2R, odata_m2d_C2R);

  hipfftHandle plan_m2d_R2C;
  size_t* work_size_m2d_R2C;
  float2* odata_m2d_R2C;
  float* idata_m2d_R2C;

  hipfftMakePlan2d(plan_m2d_R2C, 10, 20, HIPFFT_R2C, work_size_m2d_R2C);

  hipfftExecR2C(plan_m2d_R2C, idata_m2d_R2C, odata_m2d_R2C);

  hipfftHandle plan_m2d_Z2Z;
  size_t* work_size_m2d_Z2Z;
  double2* odata_m2d_Z2Z;
  double2* idata_m2d_Z2Z;

  hipfftMakePlan2d(plan_m2d_Z2Z, 10, 20, HIPFFT_Z2Z, work_size_m2d_Z2Z);

  hipfftExecZ2Z(plan_m2d_Z2Z, idata_m2d_Z2Z, odata_m2d_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_m2d_Z2D;
  size_t* work_size_m2d_Z2D;
  double* odata_m2d_Z2D;
  double2* idata_m2d_Z2D;

  hipfftMakePlan2d(plan_m2d_Z2D, 10, 20, HIPFFT_Z2D, work_size_m2d_Z2D);

  hipfftExecZ2D(plan_m2d_Z2D, idata_m2d_Z2D, odata_m2d_Z2D);

  hipfftHandle plan_m2d_D2Z;
  size_t* work_size_m2d_D2Z;
  double2* odata_m2d_D2Z;
  double* idata_m2d_D2Z;

  hipfftMakePlan2d(plan_m2d_D2Z, 10, 20, HIPFFT_D2Z, work_size_m2d_D2Z);

  hipfftExecD2Z(plan_m2d_D2Z, idata_m2d_D2Z, odata_m2d_D2Z);

  hipfftHandle plan_m3d_C2C;
  size_t* work_size_m3d_C2C;
  float2* odata_m3d_C2C;
  float2* idata_m3d_C2C;

  hipfftMakePlan3d(plan_m3d_C2C, 10, 20, 30, HIPFFT_C2C, work_size_m3d_C2C);

  hipfftExecC2C(plan_m3d_C2C, idata_m3d_C2C, odata_m3d_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_m3d_C2R;
  size_t* work_size_m3d_C2R;
  float* odata_m3d_C2R;
  float2* idata_m3d_C2R;

  hipfftMakePlan3d(plan_m3d_C2R, 10, 20, 30, HIPFFT_C2R, work_size_m3d_C2R);

  hipfftExecC2R(plan_m3d_C2R, idata_m3d_C2R, odata_m3d_C2R);

  hipfftHandle plan_m3d_R2C;
  size_t* work_size_m3d_R2C;
  float2* odata_m3d_R2C;
  float* idata_m3d_R2C;

  hipfftMakePlan3d(plan_m3d_R2C, 10, 20, 30, HIPFFT_R2C, work_size_m3d_R2C);

  hipfftExecR2C(plan_m3d_R2C, idata_m3d_R2C, odata_m3d_R2C);

  hipfftHandle plan_m3d_Z2Z;
  size_t* work_size_m3d_Z2Z;
  double2* odata_m3d_Z2Z;
  double2* idata_m3d_Z2Z;

  hipfftMakePlan3d(plan_m3d_Z2Z, 10, 20, 30, HIPFFT_Z2Z, work_size_m3d_Z2Z);

  hipfftExecZ2Z(plan_m3d_Z2Z, idata_m3d_Z2Z, odata_m3d_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_m3d_Z2D;
  size_t* work_size_m3d_Z2D;
  double* odata_m3d_Z2D;
  double2* idata_m3d_Z2D;

  hipfftMakePlan3d(plan_m3d_Z2D, 10, 20, 30, HIPFFT_Z2D, work_size_m3d_Z2D);

  hipfftExecZ2D(plan_m3d_Z2D, idata_m3d_Z2D, odata_m3d_Z2D);

  hipfftHandle plan_m3d_D2Z;
  size_t* work_size_m3d_D2Z;
  double2* odata_m3d_D2Z;
  double* idata_m3d_D2Z;

  hipfftMakePlan3d(plan_m3d_D2Z, 10, 20, 30, HIPFFT_D2Z, work_size_m3d_D2Z);

  hipfftExecD2Z(plan_m3d_D2Z, idata_m3d_D2Z, odata_m3d_D2Z);

  hipfftHandle plan_mmany_C2C;
  size_t* work_size_mmany_C2C;
  int odist_mmany_C2C;
  int ostride_mmany_C2C;
  int * onembed_mmany_C2C;
  int idist_mmany_C2C;
  int istride_mmany_C2C;
  int* inembed_mmany_C2C;
  int * n_mmany_C2C;
  float2* odata_mmany_C2C;
  float2* idata_mmany_C2C;

  hipfftMakePlanMany(plan_mmany_C2C, 3, n_mmany_C2C, inembed_mmany_C2C, istride_mmany_C2C, idist_mmany_C2C, onembed_mmany_C2C, ostride_mmany_C2C, odist_mmany_C2C, HIPFFT_C2C, 12, work_size_mmany_C2C);

  hipfftExecC2C(plan_mmany_C2C, idata_mmany_C2C, odata_mmany_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_mmany_C2R;
  size_t* work_size_mmany_C2R;
  int odist_mmany_C2R;
  int ostride_mmany_C2R;
  int * onembed_mmany_C2R;
  int idist_mmany_C2R;
  int istride_mmany_C2R;
  int* inembed_mmany_C2R;
  int * n_mmany_C2R;
  float* odata_mmany_C2R;
  float2* idata_mmany_C2R;

  hipfftMakePlanMany(plan_mmany_C2R, 3, n_mmany_C2R, inembed_mmany_C2R, istride_mmany_C2R, idist_mmany_C2R, onembed_mmany_C2R, ostride_mmany_C2R, odist_mmany_C2R, HIPFFT_C2R, 12, work_size_mmany_C2R);

  hipfftExecC2R(plan_mmany_C2R, idata_mmany_C2R, odata_mmany_C2R);

  hipfftHandle plan_mmany_R2C;
  size_t* work_size_mmany_R2C;
  int odist_mmany_R2C;
  int ostride_mmany_R2C;
  int * onembed_mmany_R2C;
  int idist_mmany_R2C;
  int istride_mmany_R2C;
  int* inembed_mmany_R2C;
  int * n_mmany_R2C;
  float2* odata_mmany_R2C;
  float* idata_mmany_R2C;

  hipfftMakePlanMany(plan_mmany_R2C, 3, n_mmany_R2C, inembed_mmany_R2C, istride_mmany_R2C, idist_mmany_R2C, onembed_mmany_R2C, ostride_mmany_R2C, odist_mmany_R2C, HIPFFT_R2C, 12, work_size_mmany_R2C);

  hipfftExecR2C(plan_mmany_R2C, idata_mmany_R2C, odata_mmany_R2C);

  hipfftHandle plan_mmany_Z2Z;
  size_t* work_size_mmany_Z2Z;
  int odist_mmany_Z2Z;
  int ostride_mmany_Z2Z;
  int * onembed_mmany_Z2Z;
  int idist_mmany_Z2Z;
  int istride_mmany_Z2Z;
  int* inembed_mmany_Z2Z;
  int * n_mmany_Z2Z;
  double2* odata_mmany_Z2Z;
  double2* idata_mmany_Z2Z;

  hipfftMakePlanMany(plan_mmany_Z2Z, 3, n_mmany_Z2Z, inembed_mmany_Z2Z, istride_mmany_Z2Z, idist_mmany_Z2Z, onembed_mmany_Z2Z, ostride_mmany_Z2Z, odist_mmany_Z2Z, HIPFFT_Z2Z, 12, work_size_mmany_Z2Z);

  hipfftExecZ2Z(plan_mmany_Z2Z, idata_mmany_Z2Z, odata_mmany_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_mmany_Z2D;
  size_t* work_size_mmany_Z2D;
  int odist_mmany_Z2D;
  int ostride_mmany_Z2D;
  int * onembed_mmany_Z2D;
  int idist_mmany_Z2D;
  int istride_mmany_Z2D;
  int* inembed_mmany_Z2D;
  int * n_mmany_Z2D;
  double* odata_mmany_Z2D;
  double2* idata_mmany_Z2D;

  hipfftMakePlanMany(plan_mmany_Z2D, 3, n_mmany_Z2D, inembed_mmany_Z2D, istride_mmany_Z2D, idist_mmany_Z2D, onembed_mmany_Z2D, ostride_mmany_Z2D, odist_mmany_Z2D, HIPFFT_Z2D, 12, work_size_mmany_Z2D);

  hipfftExecZ2D(plan_mmany_Z2D, idata_mmany_Z2D, odata_mmany_Z2D);

  hipfftHandle plan_mmany_D2Z;
  size_t* work_size_mmany_D2Z;
  int odist_mmany_D2Z;
  int ostride_mmany_D2Z;
  int * onembed_mmany_D2Z;
  int idist_mmany_D2Z;
  int istride_mmany_D2Z;
  int* inembed_mmany_D2Z;
  int * n_mmany_D2Z;
  double2* odata_mmany_D2Z;
  double* idata_mmany_D2Z;

  hipfftMakePlanMany(plan_mmany_D2Z, 3, n_mmany_D2Z, inembed_mmany_D2Z, istride_mmany_D2Z, idist_mmany_D2Z, onembed_mmany_D2Z, ostride_mmany_D2Z, odist_mmany_D2Z, HIPFFT_D2Z, 12, work_size_mmany_D2Z);

  hipfftExecD2Z(plan_mmany_D2Z, idata_mmany_D2Z, odata_mmany_D2Z);

  hipfftHandle plan_mmany64_C2C;
  size_t* work_size_mmany64_C2C;
  long long int odist_mmany64_C2C;
  long long int ostride_mmany64_C2C;
  long long int * onembed_mmany64_C2C;
  long long int idist_mmany64_C2C;
  long long int istride_mmany64_C2C;
  long long int* inembed_mmany64_C2C;
  long long int * n_mmany64_C2C;
  float2* odata_mmany64_C2C;
  float2* idata_mmany64_C2C;

  hipfftMakePlanMany64(plan_mmany64_C2C, 3, n_mmany64_C2C, inembed_mmany64_C2C, istride_mmany64_C2C, idist_mmany64_C2C, onembed_mmany64_C2C, ostride_mmany64_C2C, odist_mmany64_C2C, HIPFFT_C2C, 12, work_size_mmany64_C2C);

  hipfftExecC2C(plan_mmany64_C2C, idata_mmany64_C2C, odata_mmany64_C2C, HIPFFT_FORWARD);

  hipfftHandle plan_mmany64_C2R;
  size_t* work_size_mmany64_C2R;
  long long int odist_mmany64_C2R;
  long long int ostride_mmany64_C2R;
  long long int * onembed_mmany64_C2R;
  long long int idist_mmany64_C2R;
  long long int istride_mmany64_C2R;
  long long int* inembed_mmany64_C2R;
  long long int * n_mmany64_C2R;
  float* odata_mmany64_C2R;
  float2* idata_mmany64_C2R;

  hipfftMakePlanMany64(plan_mmany64_C2R, 3, n_mmany64_C2R, inembed_mmany64_C2R, istride_mmany64_C2R, idist_mmany64_C2R, onembed_mmany64_C2R, ostride_mmany64_C2R, odist_mmany64_C2R, HIPFFT_C2R, 12, work_size_mmany64_C2R);

  hipfftExecC2R(plan_mmany64_C2R, idata_mmany64_C2R, odata_mmany64_C2R);

  hipfftHandle plan_mmany64_R2C;
  size_t* work_size_mmany64_R2C;
  long long int odist_mmany64_R2C;
  long long int ostride_mmany64_R2C;
  long long int * onembed_mmany64_R2C;
  long long int idist_mmany64_R2C;
  long long int istride_mmany64_R2C;
  long long int* inembed_mmany64_R2C;
  long long int * n_mmany64_R2C;
  float2* odata_mmany64_R2C;
  float* idata_mmany64_R2C;

  hipfftMakePlanMany64(plan_mmany64_R2C, 3, n_mmany64_R2C, inembed_mmany64_R2C, istride_mmany64_R2C, idist_mmany64_R2C, onembed_mmany64_R2C, ostride_mmany64_R2C, odist_mmany64_R2C, HIPFFT_R2C, 12, work_size_mmany64_R2C);

  hipfftExecR2C(plan_mmany64_R2C, idata_mmany64_R2C, odata_mmany64_R2C);

  hipfftHandle plan_mmany64_Z2Z;
  size_t* work_size_mmany64_Z2Z;
  long long int odist_mmany64_Z2Z;
  long long int ostride_mmany64_Z2Z;
  long long int * onembed_mmany64_Z2Z;
  long long int idist_mmany64_Z2Z;
  long long int istride_mmany64_Z2Z;
  long long int* inembed_mmany64_Z2Z;
  long long int * n_mmany64_Z2Z;
  double2* odata_mmany64_Z2Z;
  double2* idata_mmany64_Z2Z;

  hipfftMakePlanMany64(plan_mmany64_Z2Z, 3, n_mmany64_Z2Z, inembed_mmany64_Z2Z, istride_mmany64_Z2Z, idist_mmany64_Z2Z, onembed_mmany64_Z2Z, ostride_mmany64_Z2Z, odist_mmany64_Z2Z, HIPFFT_Z2Z, 12, work_size_mmany64_Z2Z);

  hipfftExecZ2Z(plan_mmany64_Z2Z, idata_mmany64_Z2Z, odata_mmany64_Z2Z, HIPFFT_BACKWARD);

  hipfftHandle plan_mmany64_Z2D;
  size_t* work_size_mmany64_Z2D;
  long long int odist_mmany64_Z2D;
  long long int ostride_mmany64_Z2D;
  long long int * onembed_mmany64_Z2D;
  long long int idist_mmany64_Z2D;
  long long int istride_mmany64_Z2D;
  long long int* inembed_mmany64_Z2D;
  long long int * n_mmany64_Z2D;
  double* odata_mmany64_Z2D;
  double2* idata_mmany64_Z2D;

  hipfftMakePlanMany64(plan_mmany64_Z2D, 3, n_mmany64_Z2D, inembed_mmany64_Z2D, istride_mmany64_Z2D, idist_mmany64_Z2D, onembed_mmany64_Z2D, ostride_mmany64_Z2D, odist_mmany64_Z2D, HIPFFT_Z2D, 12, work_size_mmany64_Z2D);

  hipfftExecZ2D(plan_mmany64_Z2D, idata_mmany64_Z2D, odata_mmany64_Z2D);

  hipfftHandle plan_mmany64_D2Z;
  size_t* work_size_mmany64_D2Z;
  long long int odist_mmany64_D2Z;
  long long int ostride_mmany64_D2Z;
  long long int * onembed_mmany64_D2Z;
  long long int idist_mmany64_D2Z;
  long long int istride_mmany64_D2Z;
  long long int* inembed_mmany64_D2Z;
  long long int * n_mmany64_D2Z;
  double2* odata_mmany64_D2Z;
  double* idata_mmany64_D2Z;

  hipfftMakePlanMany64(plan_mmany64_D2Z, 3, n_mmany64_D2Z, inembed_mmany64_D2Z, istride_mmany64_D2Z, idist_mmany64_D2Z, onembed_mmany64_D2Z, ostride_mmany64_D2Z, odist_mmany64_D2Z, HIPFFT_D2Z, 12, work_size_mmany64_D2Z);

  hipfftExecD2Z(plan_mmany64_D2Z, idata_mmany64_D2Z, odata_mmany64_D2Z);

  return 0;
}