// ====------ cufft-func-ptr.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

static hipfftResult (*pt2CufftExec)(hipfftHandle, hipfftDoubleComplex *,
                                    double *) = &hipfftExecZ2D;

int main() {
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);

  double* odata;
  double2* idata;

  pt2CufftExec(plan1, idata, odata);

  return 0;
}

int foo1() {
  typedef hipfftResult (*Func_t)(hipfftHandle, hipfftDoubleComplex *, double *);

  static Func_t FuncPtr  = &hipfftExecZ2D;

  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);

  double* odata;
  double2* idata;

  FuncPtr(plan1, idata, odata);

  return 0;
}

int foo2() {
  using Func_t = hipfftResult (*)(hipfftHandle, hipfftDoubleComplex *, double *);

  Func_t FuncPtr2  = &hipfftExecZ2D;

  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);

  double* odata;
  double2* idata;

  FuncPtr2(plan1, idata, odata);

  return 0;
}

int foo3() {
  using Func_t = hipfftResult (*)(hipfftHandle, hipfftDoubleComplex *, double *);

  Func_t FuncPtr3;
  FuncPtr3 = &hipfftExecZ2D;

  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);

  double* odata;
  double2* idata;

  FuncPtr3(plan1, idata, odata);

  return 0;
}

int foo4() {
  hipfftResult (*FuncPtr4)(hipfftHandle, hipfftDoubleComplex *, double *);

  FuncPtr4 = &hipfftExecZ2D;

  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);

  double* odata;
  double2* idata;

  FuncPtr4(plan1, idata, odata);

  return 0;
}
