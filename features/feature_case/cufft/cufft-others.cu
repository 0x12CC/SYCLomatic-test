// ====------ cufft-others.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


int main() {
  hipfftHandle plan;
  float2* iodata;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  hipfftExecR2C(plan, (float*)iodata, iodata);

  return 0;
}

int foo2() {
  hipfftHandle plan_mmany64_Z2Z;
  size_t* work_size_mmany64_Z2Z;
  long long int odist_mmany64_Z2Z;
  long long int ostride_mmany64_Z2Z;
  long long int * onembed_mmany64_Z2Z;
  long long int idist_mmany64_Z2Z;
  long long int istride_mmany64_Z2Z;
  long long int* inembed_mmany64_Z2Z;
  long long int * n_mmany64_Z2Z;
  double2* odata_mmany64_Z2Z;
  double2* idata_mmany64_Z2Z;

  hipfftMakePlanMany64(plan_mmany64_Z2Z, 3, n_mmany64_Z2Z, inembed_mmany64_Z2Z, istride_mmany64_Z2Z, idist_mmany64_Z2Z, onembed_mmany64_Z2Z, ostride_mmany64_Z2Z, odist_mmany64_Z2Z, HIPFFT_Z2Z, 12, work_size_mmany64_Z2Z);

  hipfftExecZ2Z(plan_mmany64_Z2Z, idata_mmany64_Z2Z, odata_mmany64_Z2Z, HIPFFT_FORWARD);

  hipfftExecZ2Z(plan_mmany64_Z2Z, idata_mmany64_Z2Z, odata_mmany64_Z2Z, HIPFFT_BACKWARD);

  return 0;
}

int foo3(hipStream_t stream) {
  hipfftHandle plan;
  float2* iodata;

  hipfftSetStream(plan, stream);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  hipfftExecR2C(plan, (float*)iodata, iodata);

  return 0;
}
