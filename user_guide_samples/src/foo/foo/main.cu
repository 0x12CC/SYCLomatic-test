// ====------ main.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <stdio.h>  
#include <hip/hip_runtime.h>

extern void run_util();

__global__ void kernel_main(int n) {
  printf("kernel_main!\n"); 
}

int main(){

  kernel_main<<<1, 1>>>(1); 
  hipDeviceSynchronize();
  
  run_util();
  hipDeviceSynchronize(); 
  
  return 0;
}
